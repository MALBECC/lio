#include "hip/hip_runtime.h"
/* -*- mode: c -*- */
#include <cassert>
#include <fstream>
#include <iostream>
#include <limits>
#include <map>
#include <hip/hip_math_constants.h>
#include <string>
#include <vector>

#include "../common.h"
#include "../init.h"
#include "cuda_extra.h"
#include "../matrix.h"
#include "../timer.h"
#include "../partition.h"
#include "../scalar_vector_types.h"
#include "../global_memory_pool.h"

#include "../pointxc/calc_ggaCS.h"
#include "../pointxc/calc_ggaOS.h"
#include "../pointxc/calc_ldaCS.h"

#if USE_LIBXC
#include "../libxc/libxc_accumulate_point.h"
#endif

namespace G2G {
#if FULL_DOUBLE
texture<int2, 2, hipReadModeElementType> rmm_input_gpu_tex;
texture<int2, 2, hipReadModeElementType> rmm_input_gpu_tex2;
#else
texture<float, 2, hipReadModeElementType> rmm_input_gpu_tex;
texture<float, 2, hipReadModeElementType> rmm_input_gpu_tex2;
#endif
/** KERNELS **/
#include "gpu_variables.h"
#include "kernels/accumulate_point.h"
#include "kernels/energy.h"
#include "kernels/energy_open.h"
#include "kernels/energy_derivs.h"
#include "kernels/rmm.h"
#include "kernels/weight.h"
#include "kernels/functions.h"
#include "kernels/force.h"
#include "kernels/transpose.h"

using std::cout;
using std::vector;
using std::endl;

//extern "C" void g2g_timer_sum_start_(const char* timer_name, unsigned int length_arg);
//extern "C" void g2g_timer_sum_stop_(const char* timer_name, unsigned int length_arg);
//extern "C" void g2g_timer_sum_pause_(const char* timer_name, unsigned int length_arg);

void gpu_set_variables(void) {
  int previous_device; hipGetDevice(&previous_device);
  int gpu_devices = cudaGetGPUCount();
  for(int i = 0; i < gpu_devices; i++) {
    if(hipSetDevice(i) != hipSuccess)
      std::cout << "Error: can't set the device " << i << std::endl;
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_normalization_factor), &fortran_vars.normalization_factor, sizeof(fortran_vars.normalization_factor), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_atoms), &fortran_vars.atoms, sizeof(fortran_vars.atoms), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_Iexch), &fortran_vars.iexch, sizeof(fortran_vars.iexch), 0, hipMemcpyHostToDevice);
  }
  hipSetDevice(previous_device);
  cudaAssertNoError("set_gpu_variables");
}

template<class T> void gpu_set_atom_positions(const HostMatrix<T>& m) {
  int previous_device; hipGetDevice(&previous_device);
  int gpu_devices = cudaGetGPUCount();
  for(int i = 0; i < gpu_devices; i++) {
    if(hipSetDevice(i) != hipSuccess)
      std::cout << "Error: can't set the device " << i << std::endl;
    hipMemcpyToSymbol(HIP_SYMBOL(gpu_atom_positions), m.data, m.bytes(), 0, hipMemcpyHostToDevice);
  }
  hipSetDevice(previous_device);
}

#if FULL_DOUBLE
template void gpu_set_atom_positions<double3>(const HostMatrix<double3>& m);
#else
template void gpu_set_atom_positions<float3>(const HostMatrix<float3>& m);
#endif

template<class scalar_type>
void PointGroupGPU<scalar_type>::solve(
    Timers& timers, bool compute_rmm, bool lda, bool compute_forces, bool compute_energy,
    double& energy,double& energy_i, double& energy_c, double& energy_c1, double& energy_c2,
    HostMatrix<double>& fort_forces_ms, int inner_threads, HostMatrix<double>& rmm_output_local, bool open ){
/*
  if ( open ) {
      solve_opened( timers, compute_rmm, lda, compute_forces, compute_energy,
                    energy, energy_i, energy_c, energy_c1, energy_c2,
                    fort_forces_ms );
  }
  else {
      solve_closed( timers, compute_rmm, lda, compute_forces, compute_energy,
                    energy, fort_forces_ms, inner_threads, rmm_output_local );
  }
*/
//  counter_iter++;                                                            // For Debug FF
//  std::cout << "Grupo " << counter_iter << " Energia : " << energy << " \n"; // For Debug FF
}

template<class scalar_type>
void PointGroupGPU<scalar_type>::solve_closed(
    Timers& timers,
    bool compute_rmm, bool lda, bool compute_forces, bool compute_energy,
    double& energy,    HostMatrix<double>& fort_forces_ms,
    int inner_threads, HostMatrix<double>& rmm_output_local ){

  int device;
  hipGetDevice(&device);
  current_device = device;

  /*** Computo sobre cada cubo ****/
  CudaMatrix<scalar_type> point_weights_gpu;

  /** Compute this group's functions **/
  timers.functions.start_and_sync();
  compute_functions(compute_forces, !lda);
  timers.functions.pause_and_sync();

  uint group_m = this->total_functions();

  timers.density.start_and_sync();
  /** Load points from group **/
  HostMatrix<scalar_type> point_weights_cpu(this->number_of_points, 1);

  uint i = 0;
  for (vector<Point>::const_iterator p = this->points.begin(); p != this->points.end(); ++p, ++i) {
    point_weights_cpu(i) = p->weight;
  }

  point_weights_gpu = point_weights_cpu;

  dim3 threadBlock, threadGrid;
  /* compute density/factors */

  const int block_height= divUp(group_m, 2*DENSITY_BLOCK_SIZE);

  threadBlock = dim3(DENSITY_BLOCK_SIZE,1,1); // Hay que asegurarse que la cantidad de funciones este en rango
  threadGrid = dim3(this->number_of_points,block_height,1);

  CudaMatrix<scalar_type> partial_densities_gpu;
  CudaMatrix< vec_type<scalar_type,4> > dxyz_gpu;
  CudaMatrix< vec_type<scalar_type,4> > dd1_gpu;
  CudaMatrix< vec_type<scalar_type,4> > dd2_gpu;

  partial_densities_gpu.resize(COALESCED_DIMENSION(this->number_of_points), block_height);
  dxyz_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height);
  dd1_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height );
  dd2_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height );

#if USE_LIBXC
  CudaMatrix<scalar_type> accumulated_densities_gpu;
  CudaMatrix< vec_type<scalar_type,4> > dxyz_accum_gpu;
  CudaMatrix< vec_type<scalar_type,4> > dd1_accum_gpu;
  CudaMatrix< vec_type<scalar_type,4> > dd2_accum_gpu;

  accumulated_densities_gpu.resize(COALESCED_DIMENSION(this->number_of_points), block_height);
  dxyz_accum_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height);
  dd1_accum_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height);
  dd2_accum_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height);
#endif

  //TODO: que libxc_gpu reciba estos datos para los kernels, asi todos usan lo mismo.
  const dim3 threadGrid_accumulate(divUp(this->number_of_points,DENSITY_ACCUM_BLOCK_SIZE),1,1);
  const dim3 threadBlock_accumulate(DENSITY_ACCUM_BLOCK_SIZE,1,1);

  CudaMatrix<scalar_type> factors_gpu;
  if (compute_rmm || compute_forces) {
    factors_gpu.resize(this->number_of_points);
  }

  int transposed_width = COALESCED_DIMENSION(this->number_of_points);
  #define BLOCK_DIM 16
  dim3 transpose_grid(transposed_width / BLOCK_DIM, divUp((group_m),BLOCK_DIM), 1);
  dim3 transpose_threads(BLOCK_DIM, BLOCK_DIM, 1);

  CudaMatrix<scalar_type> function_values_transposed;
  CudaMatrix<vec_type<scalar_type,4> > gradient_values_transposed;

  // Probar si esta intercalado al pedo.
  function_values_transposed.resize(group_m, COALESCED_DIMENSION(this->number_of_points));

  if (fortran_vars.do_forces || fortran_vars.gga)
      gradient_values_transposed.resize( group_m,COALESCED_DIMENSION(this->number_of_points));

  transpose<<<transpose_grid, transpose_threads>>> (function_values_transposed.data,
      function_values.data, COALESCED_DIMENSION(this->number_of_points), group_m);

  if (fortran_vars.do_forces || fortran_vars.gga)
    transpose<<<transpose_grid, transpose_threads>>> (gradient_values_transposed.data,
        gradient_values.data, COALESCED_DIMENSION(this->number_of_points), group_m );
  // fin intercalado al pedo

  HostMatrix<scalar_type> rmm_input_cpu(COALESCED_DIMENSION(group_m), group_m+DENSITY_BLOCK_SIZE);
  get_rmm_input(rmm_input_cpu); //Achica la matriz densidad a la version reducida del grupo

  for (uint i=0; i<(group_m+DENSITY_BLOCK_SIZE); i++)
  {
    for(uint j=0; j<COALESCED_DIMENSION(group_m); j++)
    {
      if((i>=group_m) || (j>=group_m) || (j > i))
      {
        rmm_input_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=0.0f;
      }
    }
  }

  /*
   **********************************************************************
   * Pasando RDM (rmm) a texturas
   **********************************************************************
   */

  hipArray* cuArray;
  hipMallocArray(&cuArray, &rmm_input_gpu_tex.channelDesc, rmm_input_cpu.width, rmm_input_cpu.height);
  hipMemcpyToArray(cuArray, 0, 0, rmm_input_cpu.data, sizeof(scalar_type)*rmm_input_cpu.width*rmm_input_cpu.height, hipMemcpyHostToDevice);
  hipBindTextureToArray(rmm_input_gpu_tex, cuArray);

  rmm_input_gpu_tex.normalized = false;

#if USE_LIBXC
  const int nspin = XC_UNPOLARIZED;
  const int functionalExchange = fortran_vars.ex_functional_id; // 1101;
  const int functionalCorrelation = fortran_vars.ec_functional_id; // 1130;
  LibxcProxy<scalar_type,4> libxcProxy;
  if (fortran_vars.use_libxc) {
    libxcProxy.init (functionalExchange, functionalCorrelation, nspin);
  }
#endif

  if (compute_energy) {
    CudaMatrix<scalar_type> energy_gpu(this->number_of_points);

#define compute_parameters \
        energy_gpu.data, factors_gpu.data, point_weights_gpu.data, this->number_of_points, function_values_transposed.data, \
        gradient_values_transposed.data, hessian_values_transposed.data, group_m, partial_densities_gpu.data, dxyz_gpu.data, \
        dd1_gpu.data,dd2_gpu.data

#define accumulate_parameters \
        energy_gpu.data, factors_gpu.data, point_weights_gpu.data, this->number_of_points, block_height, \
        partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data

// VER QUE PASA SI SACAMOS COMPUTE_FACTOR Y COMPUTE ENERGY DE gpu_compute_density
    if (compute_forces || compute_rmm) {
      if (lda)
      {
          gpu_compute_density<scalar_type, true, true, true><<<threadGrid, threadBlock>>>(compute_parameters);
          gpu_accumulate_point<scalar_type, true, true, true><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
      }
      else
      {
          gpu_compute_density<scalar_type, true, true, false><<<threadGrid, threadBlock>>>(compute_parameters);
#if USE_LIBXC
	    if (fortran_vars.use_libxc) {
	      // Accumulate the data for libxc
	      gpu_accumulate_point_for_libxc<scalar_type, true, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
		point_weights_gpu.data, this->number_of_points, block_height,
		partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data,
		accumulated_densities_gpu.data, dxyz_accum_gpu.data, dd1_accum_gpu.data, dd2_accum_gpu.data);
	#if LIBXC_CPU
	      // Compute exc_corr and y2a with libxc CPU version.
	      libxc_exchange_correlation_cpu<scalar_type, true, true, false> (&libxcProxy,
		energy_gpu.data, factors_gpu.data, this->number_of_points,
		accumulated_densities_gpu.data, dxyz_accum_gpu.data, dd1_accum_gpu.data, dd2_accum_gpu.data);
	#else
	      // Compute exc_corr and y2a with libxc GPU version.
	      libxc_exchange_correlation_gpu<scalar_type, true, true, false> (&libxcProxy,
		energy_gpu.data, factors_gpu.data, this->number_of_points,
		accumulated_densities_gpu.data, dxyz_accum_gpu.data, dd1_accum_gpu.data, dd2_accum_gpu.data);
	#endif
	      // Merge the results.
	      gpu_accumulate_energy_and_forces_from_libxc<scalar_type, true, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
		energy_gpu.data, factors_gpu.data, point_weights_gpu.data, this->number_of_points, accumulated_densities_gpu.data);
	    } else {
              gpu_accumulate_point<scalar_type, true, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
	    }
#else
	  //print_accumulate_parameters<scalar_type> (accumulate_parameters);
          gpu_accumulate_point<scalar_type, true, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
#endif
      }
    }
    else {
      if (lda)
      {
          gpu_compute_density<scalar_type, true, false, true><<<threadGrid, threadBlock>>>(compute_parameters);
          gpu_accumulate_point<scalar_type, true, false, true><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
      }
      else
      {
          gpu_compute_density<scalar_type, true, false, false><<<threadGrid, threadBlock>>>(compute_parameters);
#if USE_LIBXC
        if (fortran_vars.use_libxc) {

	  // Accumulate the data.
	  gpu_accumulate_point_for_libxc<scalar_type, true, false, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (point_weights_gpu.data,
            this->number_of_points, block_height,
	    partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data,
	    accumulated_densities_gpu.data, dxyz_accum_gpu.data, dd1_accum_gpu.data, dd2_accum_gpu.data);

    #if LIBXC_CPU
	  // Compute exc_corr and y2a with CPU libxc.
	  libxc_exchange_correlation_cpu<scalar_type, true, false, false> (&libxcProxy,
	    energy_gpu.data, factors_gpu.data, this->number_of_points,
	    accumulated_densities_gpu.data, dxyz_accum_gpu.data, dd1_accum_gpu.data, dd2_accum_gpu.data);
    #else
	  // Compute exc_corr and y2a with libxc GPU version.
	  libxc_exchange_correlation_gpu<scalar_type, true, true, false> (&libxcProxy,
	    energy_gpu.data, factors_gpu.data, this->number_of_points,
	    accumulated_densities_gpu.data, dxyz_accum_gpu.data, dd1_accum_gpu.data, dd2_accum_gpu.data);
    #endif
	  // Merge the results.
	  gpu_accumulate_energy_and_forces_from_libxc<scalar_type, true, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
	    energy_gpu.data, factors_gpu.data, point_weights_gpu.data, this->number_of_points, accumulated_densities_gpu.data);

	} else {
          gpu_accumulate_point<scalar_type, true, false, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
        }
#else
          gpu_accumulate_point<scalar_type, true, false, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (accumulate_parameters);
#endif
      }
    }
    cudaAssertNoError("compute_density");

    HostMatrix<scalar_type> energy_cpu(energy_gpu);
    for (uint i = 0; i < this->number_of_points; i++) {
      energy += energy_cpu(i);
    }
  }
  else {
#undef compute_parameters
#undef accumulate_parameters

#define compute_parameters \
    NULL,factors_gpu.data,point_weights_gpu.data,this->number_of_points,function_values_transposed.data,gradient_values_transposed.data,hessian_values_transposed.data,group_m,partial_densities_gpu.data,dxyz_gpu.data,dd1_gpu.data,dd2_gpu.data
#define accumulate_parameters \
    NULL,factors_gpu.data,point_weights_gpu.data,this->number_of_points,block_height,partial_densities_gpu.data,dxyz_gpu.data,dd1_gpu.data,dd2_gpu.data
    if (lda)
    {
        gpu_compute_density<scalar_type, false, true, true><<<threadGrid, threadBlock>>>(compute_parameters);
        gpu_accumulate_point<scalar_type, false, true, true><<<threadGrid_accumulate, threadBlock_accumulate>>>(accumulate_parameters);
    }
    else
    {
        gpu_compute_density<scalar_type, false, true, false><<<threadGrid, threadBlock>>>(compute_parameters);
#if USE_LIBXC
        if (fortran_vars.use_libxc) {
	  // Accumulate the data.
	  gpu_accumulate_point_for_libxc<scalar_type, false, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (point_weights_gpu.data,
            this->number_of_points, block_height,
	    partial_densities_gpu.data, dxyz_gpu.data, dd1_gpu.data, dd2_gpu.data,
	    accumulated_densities_gpu.data, dxyz_accum_gpu.data, dd1_accum_gpu.data, dd2_accum_gpu.data);

    #if LIBXC_CPU
	  // Compute exc_corr and y2a with libxc CPU.
	  libxc_exchange_correlation_cpu<scalar_type, false, true, false> (&libxcProxy,
	    NULL, factors_gpu.data, this->number_of_points,
	    accumulated_densities_gpu.data, dxyz_accum_gpu.data, dd1_accum_gpu.data, dd2_accum_gpu.data);
    #else
	  // Compute exc_corr and y2a with libxc GPU version.
	  libxc_exchange_correlation_gpu<scalar_type, false, true, false> (&libxcProxy,
	    NULL, factors_gpu.data, this->number_of_points,
	    accumulated_densities_gpu.data, dxyz_accum_gpu.data, dd1_accum_gpu.data, dd2_accum_gpu.data);
    #endif
	  // Merge the results.
	  gpu_accumulate_energy_and_forces_from_libxc<scalar_type, false, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
	    NULL,factors_gpu.data, point_weights_gpu.data, this->number_of_points, accumulated_densities_gpu.data);
	} else {
    	  gpu_accumulate_point<scalar_type, false, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>>(accumulate_parameters);
	}
#else
        gpu_accumulate_point<scalar_type, false, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>>(accumulate_parameters);
#endif
    }
    cudaAssertNoError("compute_density");
  }
#undef compute_parameters
#undef accumulate_parameters

  timers.density.pause_and_sync();
  /* compute forces */
  if (compute_forces) {
    //************ Repongo los valores que puse a cero antes, para las fuerzas son necesarios (o por lo mens utiles)
    for (uint i=0; i<(group_m); i++) {
      for(uint j=0; j<(group_m); j++) {
        if((i>=group_m) || (j>=group_m) || (j > i))
        {
          rmm_input_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=rmm_input_cpu.data[COALESCED_DIMENSION(group_m)*j+i] ;
        }
      }
    }

    timers.density_derivs.start_and_sync();
    hipMemcpyToArray(cuArray, 0, 0,rmm_input_cpu.data,
      sizeof(scalar_type)*rmm_input_cpu.width*rmm_input_cpu.height, hipMemcpyHostToDevice);

    timers.density_derivs.start_and_sync();
    dim3 threads = dim3(this->number_of_points);
    threadBlock = dim3(DENSITY_DERIV_BLOCK_SIZE);
    threadGrid = divUp(threads, threadBlock);

    CudaMatrix<vec_type4> dd_gpu(COALESCED_DIMENSION(this->number_of_points), this->total_nucleii()); dd_gpu.zero();
    CudaMatrixUInt nuc_gpu(this->func2local_nuc);  // TODO: esto en realidad se podria guardar una sola vez durante su construccion

    gpu_compute_density_derivs<<<threadGrid, threadBlock>>>(
        function_values.data, gradient_values.data, nuc_gpu.data, dd_gpu.data, this->number_of_points, group_m, this->total_nucleii());
    cudaAssertNoError("density_derivs");
    timers.density_derivs.pause_and_sync();

    timers.forces.start_and_sync();
    CudaMatrix<vec_type4> forces_gpu(this->total_nucleii());

    threads = dim3(this->total_nucleii());
    threadBlock = dim3(FORCE_BLOCK_SIZE);
    threadGrid = divUp(threads, threadBlock);
    gpu_compute_forces<<<threadGrid, threadBlock>>>(
        this->number_of_points, factors_gpu.data, dd_gpu.data, forces_gpu.data, this->total_nucleii());
    cudaAssertNoError("forces");

    HostMatrix<vec_type4> forces_cpu(forces_gpu);

    for (uint i = 0; i < this->total_nucleii(); ++i) {
      vec_type4 atom_force = forces_cpu(i);
      uint global_nuc = this->local2global_nuc[i];
      fort_forces_ms(global_nuc, 0) += atom_force.x;
      fort_forces_ms(global_nuc, 1) += atom_force.y;
      fort_forces_ms(global_nuc, 2) += atom_force.z;

    }
    timers.forces.pause_and_sync();
  }

  timers.rmm.start_and_sync();
  /* compute RMM */
  if (compute_rmm) {
    threadBlock = dim3(RMM_BLOCK_SIZE_XY, RMM_BLOCK_SIZE_XY);
    uint blocksPerRow = divUp(group_m, RMM_BLOCK_SIZE_XY);
    // Only use enough blocks for lower triangle
    threadGrid = dim3(blocksPerRow*(blocksPerRow+1)/2);

    CudaMatrix<scalar_type> rmm_output_gpu(COALESCED_DIMENSION(group_m), group_m);
    rmm_output_gpu.zero();
    // For calls with a single block (pretty common with cubes) don't bother doing the arithmetic to get block position in the matrix
    if (blocksPerRow > 1) {
        gpu_update_rmm<scalar_type,true><<<threadGrid, threadBlock>>>(factors_gpu.data, this->number_of_points, rmm_output_gpu.data, function_values.data, group_m);
    } else {
        gpu_update_rmm<scalar_type,false><<<threadGrid, threadBlock>>>(factors_gpu.data, this->number_of_points, rmm_output_gpu.data, function_values.data, group_m);
    }
    cudaAssertNoError("update_rmm");

    /*** Contribute this RMM to the total RMM ***/
    HostMatrix<scalar_type> rmm_output_cpu(rmm_output_gpu);
    this->add_rmm_output(rmm_output_cpu, rmm_output_local);
  }
  timers.rmm.pause_and_sync();

  /* clear functions */
  if(!(this->inGlobal)) {
    function_values.deallocate();
    gradient_values.deallocate();
    hessian_values_transposed.deallocate();
  }
  //Deshago el bind de textura de rmm
  hipUnbindTexture(rmm_input_gpu_tex); //Enroque el Unbind con el Free, asi parece mas logico. Nano
  hipFreeArray(cuArray);

}

//======================
// OPENSHELL
//======================

template<class scalar_type>
void PointGroupGPU<scalar_type>::solve_opened(
    Timers& timers, bool compute_rmm, bool lda, bool compute_forces,
    bool compute_energy, double& energy, double& energy_i,
    double& energy_c, double& energy_c1, double& energy_c2,
    HostMatrix<double>& fort_forces_ms,
    HostMatrix<double>& rmm_output_local_a,
    HostMatrix<double>& rmm_output_local_b){

  int device;
  hipGetDevice(&device);
  current_device = device;

  /*** Computo sobre cada cubo ****/
  CudaMatrix<scalar_type> point_weights_gpu;

  /** Compute this group's functions **/
  timers.functions.start_and_sync();
  compute_functions(compute_forces, !lda);
  timers.functions.pause_and_sync();

  uint group_m = this->total_functions();

  timers.density.start_and_sync();
  /** Load points from group **/
  HostMatrix<scalar_type> point_weights_cpu(this->number_of_points, 1);

  uint i = 0;
  for (vector<Point>::const_iterator p = this->points.begin(); p != this->points.end(); ++p, ++i) {
    point_weights_cpu(i) = p->weight;
  }
  point_weights_gpu = point_weights_cpu;

  dim3 threadBlock, threadGrid;
  const int block_height= divUp(group_m,2*DENSITY_BLOCK_SIZE);

  // This makes sure the amount of functions fits within range.
  threadBlock = dim3(DENSITY_BLOCK_SIZE,1,1);
  threadGrid = dim3(this->number_of_points,block_height,1);

  CudaMatrix<scalar_type> factors_a_gpu;
  CudaMatrix<scalar_type> factors_b_gpu;

  // Gradients (dxyz) and Hessians (dd1,dd2) for alpha/beta.
  CudaMatrix<scalar_type> partial_densities_a_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dxyz_a_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd1_a_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd2_a_gpu;

  CudaMatrix<scalar_type> partial_densities_b_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dxyz_b_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd1_b_gpu;
  CudaMatrix<vec_type<scalar_type,4> > dd2_b_gpu;

  // Matrix transpose is needed for better coalescence in density.
  CudaMatrix<scalar_type> function_values_transposed;
  CudaMatrix<vec_type<scalar_type,4> > gradient_values_transposed;

  int transposed_width = COALESCED_DIMENSION(this->number_of_points);

  function_values_transposed.resize(group_m, COALESCED_DIMENSION(this->number_of_points));
  if (fortran_vars.do_forces || fortran_vars.gga)
      gradient_values_transposed.resize( group_m,COALESCED_DIMENSION(this->number_of_points));

  #define BLOCK_DIM 16
  dim3 transpose_grid(transposed_width / BLOCK_DIM, divUp((group_m),BLOCK_DIM));
  dim3 transpose_threads(BLOCK_DIM, BLOCK_DIM, 1);

  transpose<<<transpose_grid, transpose_threads>>> (function_values_transposed.data, function_values.data,  COALESCED_DIMENSION(this->number_of_points),group_m   );
  if (fortran_vars.do_forces || fortran_vars.gga)
      transpose<<<transpose_grid, transpose_threads>>> (gradient_values_transposed.data, gradient_values.data, COALESCED_DIMENSION(this->number_of_points), group_m );

  partial_densities_a_gpu.resize(COALESCED_DIMENSION(this->number_of_points), block_height);
  dxyz_a_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height);
  dd1_a_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height );
  dd2_a_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height );

  partial_densities_b_gpu.resize(COALESCED_DIMENSION(this->number_of_points), block_height);
  dxyz_b_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height);
  dd1_b_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height );
  dd2_b_gpu.resize(COALESCED_DIMENSION(this->number_of_points),block_height );

  const dim3 threadGrid_accumulate(divUp(this->number_of_points,DENSITY_ACCUM_BLOCK_SIZE),1,1);
  const dim3 threadBlock_accumulate(DENSITY_ACCUM_BLOCK_SIZE,1,1);

  if (compute_rmm || compute_forces) {
    factors_a_gpu.resize(this->number_of_points);
    factors_b_gpu.resize(this->number_of_points);
  }

  HostMatrix<scalar_type> rmm_input_a_cpu(COALESCED_DIMENSION(group_m), group_m+DENSITY_BLOCK_SIZE);
  HostMatrix<scalar_type> rmm_input_b_cpu(COALESCED_DIMENSION(group_m), group_m+DENSITY_BLOCK_SIZE);
   //Reduces density matrixes (Up,Down) to the reduced group version
  get_rmm_input(rmm_input_a_cpu, rmm_input_b_cpu);

  for (uint i=0; i<(group_m+DENSITY_BLOCK_SIZE); i++) {
    for(uint j=0; j<COALESCED_DIMENSION(group_m); j++) {
      if((i>=group_m) || (j>=group_m) || (j > i)) {
        rmm_input_a_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=0.0f;
        rmm_input_b_cpu.data[COALESCED_DIMENSION(group_m)*i+j]=0.0f;
      }
    }
  }

  /*
  **********************************************************************
  * Pasando RDM (rmm) a texturas/
  **********************************************************************
  */

  hipArray* cuArray1;
  hipArray* cuArray2;
  hipMallocArray(&cuArray1, &rmm_input_gpu_tex.channelDesc, rmm_input_a_cpu.width,rmm_input_a_cpu.height);
  hipMallocArray(&cuArray2, &rmm_input_gpu_tex2.channelDesc, rmm_input_b_cpu.width,rmm_input_b_cpu.height);
  hipMemcpyToArray(cuArray1, 0, 0,rmm_input_a_cpu.data,sizeof(scalar_type)*rmm_input_a_cpu.width*rmm_input_a_cpu.height, hipMemcpyHostToDevice);
  hipMemcpyToArray(cuArray2, 0, 0,rmm_input_b_cpu.data,sizeof(scalar_type)*rmm_input_b_cpu.width*rmm_input_b_cpu.height, hipMemcpyHostToDevice);
  hipBindTextureToArray(rmm_input_gpu_tex, cuArray1);
  hipBindTextureToArray(rmm_input_gpu_tex2, cuArray2);

  rmm_input_gpu_tex.normalized = false;
  rmm_input_gpu_tex2.normalized = false;

  if (compute_energy) {
    CudaMatrix<scalar_type> energy_gpu(this->number_of_points);
    CudaMatrix<scalar_type> energy_i_gpu(this->number_of_points);
    CudaMatrix<scalar_type> energy_c_gpu(this->number_of_points);
    CudaMatrix<scalar_type> energy_c1_gpu(this->number_of_points);
    CudaMatrix<scalar_type> energy_c2_gpu(this->number_of_points);

    if (compute_forces || compute_rmm) {
      gpu_compute_density_opened<scalar_type, true, true, false><<<threadGrid, threadBlock>>>(
             point_weights_gpu.data,this->number_of_points, function_values_transposed.data,
             gradient_values_transposed.data,hessian_values_transposed.data, group_m,
             partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
             partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
      gpu_accumulate_point_open<scalar_type, true, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
             energy_gpu.data,energy_i_gpu.data,energy_c_gpu.data,energy_c1_gpu.data,energy_c2_gpu.data,
             factors_a_gpu.data, factors_b_gpu.data, point_weights_gpu.data,this->number_of_points,block_height,
             partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
             partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
    }
    else {
      gpu_compute_density_opened<scalar_type, true, false, false><<<threadGrid, threadBlock>>>(
             point_weights_gpu.data,this->number_of_points, function_values_transposed.data,
             gradient_values_transposed.data,hessian_values_transposed.data, group_m,
             partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
             partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
      gpu_accumulate_point_open<scalar_type, true, false, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
             energy_gpu.data, energy_i_gpu.data,energy_c_gpu.data,energy_c1_gpu.data,energy_c2_gpu.data,
             factors_a_gpu.data, factors_b_gpu.data, point_weights_gpu.data,this->number_of_points,block_height,
             partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
             partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
    }
    cudaAssertNoError("compute_density");

    HostMatrix<scalar_type> energy_cpu(energy_gpu);
    HostMatrix<scalar_type> energy_i_cpu(energy_i_gpu);
    HostMatrix<scalar_type> energy_c_cpu(energy_c_gpu);
    HostMatrix<scalar_type> energy_c1_cpu(energy_c1_gpu);
    HostMatrix<scalar_type> energy_c2_cpu(energy_c2_gpu);

    for (uint i = 0; i < this->number_of_points; i++) {
      energy    += energy_cpu(i);
      energy_i  += energy_i_cpu(i);
      energy_c  += energy_c_cpu(i);
      energy_c1 += energy_c1_cpu(i);
      energy_c2 += energy_c2_cpu(i);
    }
  }
  else {
    gpu_compute_density_opened<scalar_type, false, true, false><<<threadGrid, threadBlock>>>(
           point_weights_gpu.data, this->number_of_points, function_values_transposed.data,
           gradient_values_transposed.data,hessian_values_transposed.data, group_m,
           partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
           partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
    gpu_accumulate_point_open<scalar_type, false, true, false><<<threadGrid_accumulate, threadBlock_accumulate>>> (
           NULL,NULL,NULL,NULL,NULL,
           factors_a_gpu.data, factors_b_gpu.data, point_weights_gpu.data,this->number_of_points,block_height,
           partial_densities_a_gpu.data, dxyz_a_gpu.data, dd1_a_gpu.data, dd2_a_gpu.data,
           partial_densities_b_gpu.data, dxyz_b_gpu.data, dd1_b_gpu.data, dd2_b_gpu.data);
    cudaAssertNoError("compute_density");
  }

  timers.density.pause_and_sync();


  /* compute forces */
  if (compute_forces) {

    // Repongo los valores que puse a cero antes, para las fuerzas son necesarios (o por lo menos utiles)
    for (uint i=0; i<(group_m); i++) {
    for (uint j=0; j<(group_m); j++) {
      if((i>=group_m) || (j>=group_m) || (j > i)){
        rmm_input_a_cpu.data[COALESCED_DIMENSION(group_m)*i+j] =
                        rmm_input_a_cpu.data[COALESCED_DIMENSION(group_m)*j+i] ;
        rmm_input_b_cpu.data[COALESCED_DIMENSION(group_m)*i+j] =
                        rmm_input_b_cpu.data[COALESCED_DIMENSION(group_m)*j+i] ;
      }
    }
    }

    hipMemcpyToArray(cuArray1, 0, 0,rmm_input_a_cpu.data,sizeof(scalar_type)*rmm_input_a_cpu.width*rmm_input_a_cpu.height, hipMemcpyHostToDevice);
    hipMemcpyToArray(cuArray2, 0, 0,rmm_input_b_cpu.data,sizeof(scalar_type)*rmm_input_b_cpu.width*rmm_input_b_cpu.height, hipMemcpyHostToDevice);


    dim3 threads;
    timers.density_derivs.start_and_sync();
    threads = dim3(this->number_of_points);
    threadBlock = dim3(DENSITY_DERIV_BLOCK_SIZE);
    threadGrid = divUp(threads, threadBlock);

    CudaMatrix<vec_type4> dd_gpu_a(COALESCED_DIMENSION(this->number_of_points), this->total_nucleii());
    CudaMatrix<vec_type4> dd_gpu_b(COALESCED_DIMENSION(this->number_of_points), this->total_nucleii());
    dd_gpu_a.zero();
    dd_gpu_b.zero();
    CudaMatrixUInt nuc_gpu(this->func2local_nuc);

    // Kernel
    gpu_compute_density_derivs_open<<<threadGrid, threadBlock>>>(function_values.data, gradient_values.data, nuc_gpu.data, dd_gpu_a.data, dd_gpu_b.data, this->number_of_points, group_m, this->total_nucleii());

    cudaAssertNoError("density_derivs");
    timers.density_derivs.pause_and_sync();

    timers.forces.start_and_sync();
    CudaMatrix<vec_type4> forces_gpu_a(this->total_nucleii());
    CudaMatrix<vec_type4> forces_gpu_b(this->total_nucleii());

    threads = dim3(this->total_nucleii());
    threadBlock = dim3(FORCE_BLOCK_SIZE);
    threadGrid = divUp(threads, threadBlock);
    // Kernel
    gpu_compute_forces<<<threadGrid, threadBlock>>>(this->number_of_points, factors_a_gpu.data, dd_gpu_a.data, forces_gpu_a.data, this->total_nucleii());
    gpu_compute_forces<<<threadGrid, threadBlock>>>(this->number_of_points, factors_b_gpu.data, dd_gpu_b.data, forces_gpu_b.data, this->total_nucleii());

    cudaAssertNoError("forces");

    HostMatrix<vec_type4> forces_cpu_a(forces_gpu_a);
    HostMatrix<vec_type4> forces_cpu_b(forces_gpu_b);

    for (uint i = 0; i < this->total_nucleii(); ++i) {
      vec_type4 atom_force_a = forces_cpu_a(i);
      vec_type4 atom_force_b = forces_cpu_b(i);
      uint global_nuc = this->local2global_nuc[i];

      fort_forces_ms(global_nuc, 0) += atom_force_a.x + atom_force_b.x;
      fort_forces_ms(global_nuc, 1) += atom_force_a.y + atom_force_b.y;
      fort_forces_ms(global_nuc, 2) += atom_force_a.z + atom_force_b.z;
    }

    timers.forces.pause_and_sync();
  }

  /* compute RMM */
  timers.rmm.start_and_sync();
  if (compute_rmm) {
    threadBlock = dim3(RMM_BLOCK_SIZE_XY, RMM_BLOCK_SIZE_XY);
    uint blocksPerRow = divUp(group_m, RMM_BLOCK_SIZE_XY);
    // Only use enough blocks for lower triangle
    threadGrid = dim3(blocksPerRow*(blocksPerRow+1)/2);

    CudaMatrix<scalar_type> rmm_output_a_gpu(COALESCED_DIMENSION(group_m), group_m);
    CudaMatrix<scalar_type> rmm_output_b_gpu(COALESCED_DIMENSION(group_m), group_m);
    //rmm_output_a_gpu.zero();
    //rmm_output_b_gpu.zero();
    // For calls with a single block (pretty common with cubes) don't bother doing the arithmetic to get block position in the matrix
    if (blocksPerRow > 1) {
        gpu_update_rmm<scalar_type,true><<<threadGrid, threadBlock>>>(factors_a_gpu.data, this->number_of_points, rmm_output_a_gpu.data, function_values.data, group_m);
        gpu_update_rmm<scalar_type,true><<<threadGrid, threadBlock>>>(factors_b_gpu.data, this->number_of_points, rmm_output_b_gpu.data, function_values.data, group_m);
    } else {
        gpu_update_rmm<scalar_type,false><<<threadGrid, threadBlock>>>(factors_a_gpu.data, this->number_of_points, rmm_output_a_gpu.data, function_values.data, group_m);
        gpu_update_rmm<scalar_type,false><<<threadGrid, threadBlock>>>(factors_b_gpu.data, this->number_of_points, rmm_output_b_gpu.data, function_values.data, group_m);
    }
    cudaAssertNoError("update_rmm");
    /*** Contribute this RMM to the total RMM ***/
    HostMatrix<scalar_type> rmm_output_a_cpu(rmm_output_a_gpu);
    HostMatrix<scalar_type> rmm_output_b_cpu(rmm_output_b_gpu);
    this->add_rmm_output(rmm_output_a_cpu, rmm_output_local_a);
    this->add_rmm_output(rmm_output_b_cpu, rmm_output_local_b);
  }
  timers.rmm.pause_and_sync();

  /* clear functions */
  if(!(this->inGlobal)) {
    function_values.deallocate();
    gradient_values.deallocate();
    hessian_values_transposed.deallocate();
  }

  //Deshago el bind de textura de rmm
  hipUnbindTexture(rmm_input_gpu_tex); //Enroque el Unbind con el Free, asi parece mas logico. Nano
  hipUnbindTexture(rmm_input_gpu_tex2); //Enroque el Unbind con el Free, asi parece mas logico. Nano
  hipFreeArray(cuArray1);
  hipFreeArray(cuArray2);

  //uint free_memory, total_memory;
  //cudaGetMemoryInfo(free_memory, total_memory);
  //cout << "Maximum used memory: " << (double)max_used_memory / (1024 * 1024) << "MB (" << ((double)max_used_memory / total_memory) * 100.0 << "%)" << endl;
  //cudaPrintMemoryInfo();
}



/*******************************
 * Cube Functions
 *******************************/

template<class scalar_type>
void PointGroupGPU<scalar_type>::compute_functions(bool forces, bool gga)
{
  if(this->inGlobal) //Ya las tengo en memoria? entonces salgo porque ya estan las 3 calculadas
    return;

  if(0 == GlobalMemoryPool::tryAlloc(this->size_in_gpu())) //1 si hubo error, 0 si pude reservar la memoria
    this->inGlobal=true;
  CudaMatrix<vec_type4> points_position_gpu;
  CudaMatrix<vec_type2> factor_ac_gpu;
  CudaMatrixUInt nuc_gpu;
  CudaMatrixUInt contractions_gpu;

  /** Load points from group **/
  {
    HostMatrix<vec_type4> points_position_cpu(this->number_of_points, 1);
    uint i = 0;
    for (vector<Point>::const_iterator p = this->points.begin(); p != this->points.end(); ++p, ++i) {
      points_position_cpu(i) = vec_type4(p->position.x, p->position.y, p->position.z, 0);
    }
    points_position_gpu = points_position_cpu;
  }
  /* Load group functions */
  uint group_m = this->s_functions + this->p_functions * 3 + this->d_functions * 6;
  uint4 group_functions = make_uint4(this->s_functions, this->p_functions, this->d_functions, group_m);
  HostMatrix<vec_type2> factor_ac_cpu(COALESCED_DIMENSION(group_m), MAX_CONTRACTIONS);
  HostMatrixUInt nuc_cpu(group_m, 1), contractions_cpu(group_m, 1);

  // TODO: hacer que functions.h itere por total_small_functions()... asi puedo hacer que
  // func2global_nuc sea de tamaño total_functions() y directamente copio esa matriz aca y en otros lados

  uint ii = 0;
  for (uint i = 0; i < this->total_functions_simple(); ++i) {
    uint inc = this->small_function_type(i);

    uint func = this->local2global_func[i];
    uint this_nuc = this->func2global_nuc(i);
    uint this_cont = fortran_vars.contractions(func);

    for (uint j = 0; j < inc; j++) {
      nuc_cpu(ii) = this_nuc;
      contractions_cpu(ii) = this_cont;
      for (unsigned int k = 0; k < this_cont; k++)
        factor_ac_cpu(ii, k) = vec_type2(fortran_vars.a_values(func, k), fortran_vars.c_values(func, k));
      ii++;
    }
  }
  factor_ac_gpu = factor_ac_cpu;
  nuc_gpu = nuc_cpu;
  contractions_gpu = contractions_cpu;

  CudaMatrix<vec_type<scalar_type,4> > hessian_values;
  /** Compute Functions **/
  function_values.resize(COALESCED_DIMENSION(this->number_of_points), group_functions.w);
  if (fortran_vars.do_forces || fortran_vars.gga)
      gradient_values.resize(COALESCED_DIMENSION(this->number_of_points), group_functions.w);
  if (fortran_vars.gga)
      hessian_values.resize(COALESCED_DIMENSION(this->number_of_points), (group_functions.w) * 2);

  dim3 threads(this->number_of_points);
  dim3 threadBlock(FUNCTIONS_BLOCK_SIZE);
  dim3 threadGrid = divUp(threads, threadBlock);

#define compute_functions_parameters \
  points_position_gpu.data,this->number_of_points,contractions_gpu.data,factor_ac_gpu.data,nuc_gpu.data,function_values.data,gradient_values.data,hessian_values.data,group_functions
  if (forces) {
    if (gga)
      gpu_compute_functions<scalar_type, true, true><<<threadGrid, threadBlock>>>(compute_functions_parameters);
    else
      gpu_compute_functions<scalar_type, true, false><<<threadGrid, threadBlock>>>(compute_functions_parameters);
  }
  else {
    if (gga)
      gpu_compute_functions<scalar_type, false, true><<<threadGrid, threadBlock>>>(compute_functions_parameters);
    else
      gpu_compute_functions<scalar_type, false, false><<<threadGrid, threadBlock>>>(compute_functions_parameters);
  }

  if (fortran_vars.gga) {
    int transposed_width = COALESCED_DIMENSION(this->number_of_points);
    #define BLOCK_DIM 16
    dim3 transpose_threads(BLOCK_DIM, BLOCK_DIM, 1);
    dim3 transpose_grid=dim3(transposed_width / BLOCK_DIM, divUp((group_m)*2, BLOCK_DIM), 1);

    hessian_values_transposed.resize((group_m) * 2, COALESCED_DIMENSION(this->number_of_points));
    transpose<<<transpose_grid, transpose_threads>>> (hessian_values_transposed.data,
        hessian_values.data, COALESCED_DIMENSION(this->number_of_points), (group_m)*2);
  }
  cudaAssertNoError("compute_functions");
}

/*******************************
 * Cube Weights
 *******************************/
template<class scalar_type>
void PointGroupGPU<scalar_type>::compute_weights(void)
{
  CudaMatrix<vec_type4> point_positions_gpu;
  CudaMatrix<vec_type4> atom_position_rm_gpu;
  {
    HostMatrix<vec_type4> points_positions_cpu(this->number_of_points, 1);
		uint i = 0;
		for (vector<Point>::const_iterator p = this->points.begin(); p != this->points.end(); ++p, ++i) {
			points_positions_cpu(i) = vec_type4(p->position.x, p->position.y, p->position.z, p->atom);
		}
    point_positions_gpu = points_positions_cpu;

    HostMatrix<vec_type4> atom_position_rm_cpu(fortran_vars.atoms, 1);
    for (uint i = 0; i < fortran_vars.atoms; i++) {
      double3 atom_pos = fortran_vars.atom_positions(i);
      atom_position_rm_cpu(i) = vec_type4(atom_pos.x, atom_pos.y, atom_pos.z, fortran_vars.rm(i));
    }
    atom_position_rm_gpu = atom_position_rm_cpu;
  }

  CudaMatrixUInt nucleii_gpu(this->local2global_nuc);

  CudaMatrix<scalar_type> weights_gpu(this->number_of_points);
  dim3 threads(this->number_of_points);
  dim3 blockSize(WEIGHT_BLOCK_SIZE);
  dim3 gridSize = divUp(threads, blockSize);
  gpu_compute_weights<scalar_type><<<gridSize,blockSize>>>(
      this->number_of_points, point_positions_gpu.data, atom_position_rm_gpu.data, weights_gpu.data, nucleii_gpu.data, this->total_nucleii());
  cudaAssertNoError("compute_weights");

  HostMatrix<scalar_type> weights_cpu(weights_gpu);
  uint i = 0;
  for (vector<Point>::iterator p =this->points.begin(); p != this->points.end(); ++p, ++i) {
    p->weight *= weights_cpu(i);
    }
}

#if FULL_DOUBLE
template class PointGroup<double>;
template class PointGroupGPU<double>;
#else
template class PointGroup<float>;
template class PointGroupGPU<float>;
#endif

}
