//#ifndef GPU_KENERLS
//#define GPU_KERNELS
//#endif

#include <cassert>
#include <fstream>
#include <iostream>
#include <limits>
#include <map>
#include <hip/hip_math_constants.h>
#include <float.h>
#include <string>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include "../../../g2g/common.h"
#include "../../../g2g/init.h"
#include "../../../g2g/cuda/cuda_extra.h"
#include "../../../g2g/matrix.h"
#include "../../../g2g/timer.h"
#include "../../../g2g/partition.h"
#include "../../../g2g/scalar_vector_types.h"
#include "../../../g2g/global_memory_pool.h"

#include "../../../g2g/libxc/libxcproxy.h"
#include "../../../g2g/libxc/libxc_accumulate_point.h"
#include "../../unit_test/commons/test_input.h"

//////////////////////////////////////
//// CALC_GGA
#define POT_ALPHA     ((scalar_type)-0.738558766382022447)
#define POT_GL        ((scalar_type)0.620350490899400087)

#define POT_VOSKO_A1  ((scalar_type)0.03109205)
#define POT_VOSKO_B1  ((scalar_type)3.72744)
#define POT_VOSKO_C1  ((scalar_type)12.9352)
#define POT_VOSKO_X0  ((scalar_type)-0.10498)
#define POT_VOSKO_Q   ((scalar_type)6.15199066246304849)
#define POT_VOSKO_A16 ((scalar_type)0.005182008333)
#define POT_VOSKO_Q2  ((scalar_type)4.7309269)

#define POT_ALYP  ((scalar_type)0.04918)
#define POT_BLYP  ((scalar_type)0.132)
#define POT_CLYP  ((scalar_type)0.2533)
#define POT_CLYP3 ((scalar_type)0.0844333333)
#define POT_DLYP  ((scalar_type)0.349)
#define POT_DLYP3 ((scalar_type)0.116333333)
#define POT_CF    ((scalar_type)2.87123400018819)
#define POT_BETA  ((scalar_type)0.0042)

#define POT_ALF ((scalar_type)0.023266)
#define POT_BET ((scalar_type)7.389)
#define POT_GAM ((scalar_type)8.723)
#define POT_DEL ((scalar_type)0.472)

extern "C" void g2g_timer_sum_start_(const char* timer_name, unsigned int length_arg);
extern "C" void g2g_timer_sum_stop_(const char* timer_name, unsigned int length_arg);
extern "C" void g2g_timer_sum_pause_(const char* timer_name, unsigned int length_arg);

///////////////////////////////////////////////////
// Cuda Matrix Helpers
//
G2G::HostMatrix< G2G::vec_type<double,4> > createMatrixD(int size)
{
    G2G::HostMatrix< G2G::vec_type<double,4> > aHostMatrix(size, 1);
    G2G::vec_type<float,4> zero(0,0,0,0);

    for (int i=0; i<5; i++){
	aHostMatrix(i).x = zero.x;
	aHostMatrix(i).y = zero.y;
	aHostMatrix(i).z = zero.z;
    }

    return aHostMatrix;
}

G2G::HostMatrix< G2G::vec_type<float,4> > createMatrixF(int size)
{
    G2G::HostMatrix< G2G::vec_type<float,4> > aHostMatrix(size, 1);
    G2G::vec_type<float,4> zero(0,0,0,0);

    for (int i=0; i<5; i++){
	aHostMatrix(i).x = zero.x;
	aHostMatrix(i).y = zero.y;
	aHostMatrix(i).z = zero.z;
    }

    return aHostMatrix;
}


//////////////////////////////////////
//// TESTS

//////////////////////////////////////////////////////////////////
// Test: iteration_test0007()
//
// Simulamos una iteracion del core de lio
// para el caso el tipo de datos Float y
// el template con los valores
// <true, true, false>
//
void iteration_test0007() {
    printf("accumulate_data_for_libxc_test0007() DOUBLES\n");
    hipError_t err = hipSuccess;

    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_in = createMatrixD(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_in = createMatrixD(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_in = createMatrixD(number_of_points);

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<double,4> > dxyz_gpu_accum = createMatrixD(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd1_gpu_accum = createMatrixD(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<double,4> > dd2_gpu_accum = createMatrixD(number_of_points);

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),1);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),1);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),1);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),1);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),1);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),1);

    // Now the arrays for energy, factors, point_weight and partial_density
    double *point_weights_gpu_in = NULL;
    double *partial_density_gpu_in = NULL;
    // Accum
    double *partial_density_gpu_accum = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(double);
    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_accum, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Now the arrays for energy, factors
    double *energy_gpu_in = NULL;
    double *factor_gpu_in = NULL;
    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    // Launch the CUDA Kernel
    int numElements = n+m;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    ///////////////////////
    // Set data
    double partial_densities_cpu[10] = {1.016692e-33,2.333626e-34,8.367814e-34,6.744978e-35,4.493371e-36,4.396106e-37,1.908333e-34,4.848228e-35,7.228556e-34,1.717567e-38};
    double point_weights_cpu[10] = {0.000000e+00,0.000000e+00,6.356219e-06,3.324887e-04,3.143648e-02,3.212402e-01,1.299464e-05,7.277725e-04,0.000000e+00,2.066700e+00};

    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemcpy(point_weights_gpu_in, point_weights_cpu, size, hipMemcpyHostToDevice);
    hipMemcpy(partial_density_gpu_in, partial_densities_cpu, size, hipMemcpyHostToDevice);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    /////////////////////////////////
    // LIBXC VERSION
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    // ACCUMULATE DATA
    gpu_accumulate_point_for_libxc<double,true,true,false><<<blocksPerGrid,threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in.data, dd1_gpu_in.data, dd2_gpu_in.data,
	partial_density_gpu_accum, dxyz_gpu_accum.data, dd1_gpu_accum.data, dd2_gpu_accum.data);

    // EXCHANGE_CORRELATION WITH LIBXC_CUDA
    // Calculate exc_corr and y2a

    libxc_exchange_correlation_gpu<double, true, true, false> (&libxcProxy,
	energy_gpu_in,
	factor_gpu_in,
	number_of_points,
	partial_density_gpu_accum,
	dxyz_gpu_accum.data,
        dd1_gpu_accum.data,
	dd2_gpu_accum.data);

    // MERGE RESULTS.
    // Join the results.
    gpu_accumulate_energy_and_forces_from_libxc<double, true, true, false><<<blocksPerGrid, threadsPerBlock>>> (
	energy_gpu_in,
	factor_gpu_in,
	point_weights_gpu_in,
	number_of_points,
	partial_density_gpu_accum);

    ///////////////////////////
    // Print libxc results
    // Allocate the host input vectors
    double *energy_cpu = (double *)malloc(size);
    double *factor_cpu = (double *)malloc(size);

    // Copy the vectors from gpu to cpu
    // Be aware that energy_gpu can be NULL.
    err = hipMemcpy(energy_cpu, energy_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector energy_gpu_in from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(factor_cpu, factor_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector factor_gpu_in from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (NULL, NULL, NULL, energy_cpu,
	factor_cpu, NULL, NULL, number_of_points);

    ////////////////////////////
    // Free Memory CPU
    free(energy_cpu);
    free(factor_cpu);

    ///////////////////////////
    // Free memory GPU
    hipFree (point_weights_gpu_in);
    hipFree (partial_density_gpu_in);
    hipFree (partial_density_gpu_accum);
    hipFree (energy_gpu_in);
    hipFree (factor_gpu_in);

}


//////////////////////////////////////////////////////////////////
// Test: iteration_test0008()
//
// Simulamos una iteracion del core de lio
// para el caso el tipo de datos Float y
// el template con los valores
// <true, true, false>
//
void iteration_test0008() {
    printf("accumulate_data_for_libxc_test0008() FLOATS\n");
    hipError_t err = hipSuccess;

    uint n = 5;
    uint m = 5;
    uint number_of_points = n+m;

    // Input
    G2G::CudaMatrix< G2G::vec_type<float,4> > dxyz_gpu_in = createMatrixF(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<float,4> > dd1_gpu_in = createMatrixF(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<float,4> > dd2_gpu_in = createMatrixF(number_of_points);

    // Accumulate
    G2G::CudaMatrix< G2G::vec_type<float,4> > dxyz_gpu_accum = createMatrixF(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<float,4> > dd1_gpu_accum = createMatrixF(number_of_points);
    G2G::CudaMatrix< G2G::vec_type<float,4> > dd2_gpu_accum = createMatrixF(number_of_points);

    dxyz_gpu_in.resize(COALESCED_DIMENSION(number_of_points),1);
    dd1_gpu_in.resize(COALESCED_DIMENSION(number_of_points),1);
    dd2_gpu_in.resize(COALESCED_DIMENSION(number_of_points),1);

    dxyz_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),1);
    dd1_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),1);
    dd2_gpu_accum.resize(COALESCED_DIMENSION(number_of_points),1);

    // Now the arrays for energy, factors, point_weight and partial_density
    float *point_weights_gpu_in = NULL;
    float *partial_density_gpu_in = NULL;
    // Accum
    float *partial_density_gpu_accum = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(float);
    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_accum, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Now the arrays for energy, factors
    float *energy_gpu_in = NULL;
    float *factor_gpu_in = NULL;
    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    // Launch the CUDA Kernel
    int numElements = n+m;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    ///////////////////////
    // Set data
    float partial_densities_cpu[10] = {1.016692e-33,2.333626e-34,8.367814e-34,6.744978e-35,4.493371e-36,4.396106e-37,1.908333e-34,4.848228e-35,7.228556e-34,1.717567e-38};
    float point_weights_cpu[10] = {0.000000e+00,0.000000e+00,6.356219e-06,3.324887e-04,3.143648e-02,3.212402e-01,1.299464e-05,7.277725e-04,0.000000e+00,2.066700e+00};

    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemcpy(point_weights_gpu_in, point_weights_cpu, size, hipMemcpyHostToDevice);
    hipMemcpy(partial_density_gpu_in, partial_densities_cpu, size, hipMemcpyHostToDevice);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<float,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    /////////////////////////////////
    // LIBXC VERSION
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    // ACCUMULATE DATA
    gpu_accumulate_point_for_libxc<float,true,true,false><<<blocksPerGrid,threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in.data, dd1_gpu_in.data, dd2_gpu_in.data,
	partial_density_gpu_accum, dxyz_gpu_accum.data, dd1_gpu_accum.data, dd2_gpu_accum.data);

    // EXCHANGE_CORRELATION WITH LIBXC_CUDA
    // Calculate exc_corr and y2a
    libxc_exchange_correlation_gpu<float, true, true, false> (&libxcProxy,
	energy_gpu_in,
	factor_gpu_in,
	number_of_points,
	partial_density_gpu_accum,
	dxyz_gpu_accum.data,
        dd1_gpu_accum.data,
	dd2_gpu_accum.data);
    
    // MERGE RESULTS.
    // Join the results.
    gpu_accumulate_energy_and_forces_from_libxc<float, true, true, false><<<blocksPerGrid, threadsPerBlock>>> (
	energy_gpu_in,
	factor_gpu_in,
	point_weights_gpu_in,
	number_of_points,
	partial_density_gpu_accum);

    ///////////////////////////
    // Print libxc results
    // Allocate the host input vectors
    float *energy_cpu = (float *)malloc(size);
    float *factor_cpu = (float *)malloc(size);

    // Copy the vectors from gpu to cpu
    // Be aware that energy_gpu can be NULL.
    err = hipMemcpy(energy_cpu, energy_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector energy_gpu_in from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(factor_cpu, factor_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector factor_gpu_in from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (NULL, NULL, NULL, energy_cpu,
	factor_cpu, NULL, NULL, number_of_points);

    ////////////////////////////
    // Free Memory CPU
    free(energy_cpu);
    free(factor_cpu);

    ///////////////////////////
    // Free memory GPU
    hipFree (point_weights_gpu_in);
    hipFree (partial_density_gpu_in);
    hipFree (partial_density_gpu_accum);
    hipFree (energy_gpu_in);
    hipFree (factor_gpu_in);

}


//////////////////////////////////////////////////////////////////
// Test: iteration_test0009()
//
// Simulamos una iteracion del core de lio
// para el caso el tipo de datos Float y
// el template con los valores
// <true, true, false>
//
void iteration_test0009() {
    printf("accumulate_data_for_libxc_test0008() FLOATS\n");
    hipError_t err = hipSuccess;

    uint number_of_points = 221;
    uint vec_size = sizeof(G2G::vec_type<float,4>)*number_of_points;

    // Input
    G2G::vec_type<float,4>* dxyz_gpu_in = NULL;
    G2G::vec_type<float,4>* dd1_gpu_in = NULL;
    G2G::vec_type<float,4>* dd2_gpu_in = NULL;

    // Accumulate
    G2G::vec_type<float,4>* dxyz_gpu_accum = NULL;
    G2G::vec_type<float,4>* dd1_gpu_accum = NULL;
    G2G::vec_type<float,4>* dd2_gpu_accum = NULL;

    // Alloc memory for the arrays.
    hipMalloc ((void**)&dxyz_gpu_in, vec_size);
    hipMalloc ((void**)&dd1_gpu_in, vec_size);
    hipMalloc ((void**)&dd2_gpu_in, vec_size);

    hipMalloc ((void**)&dxyz_gpu_accum, vec_size);
    hipMalloc ((void**)&dd1_gpu_accum, vec_size);
    hipMalloc ((void**)&dd2_gpu_accum, vec_size);


    // Now the arrays for energy, factors, point_weight and partial_density
    float *point_weights_gpu_in = NULL;
    float *partial_density_gpu_in = NULL;
    float *partial_density_gpu_accum = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(float);
    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_accum, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Now the arrays for energy, factors
    float *energy_gpu_in = NULL;
    float *factor_gpu_in = NULL;
    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    // Launch the CUDA Kernel
    int numElements = 221;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    ///////////////////////
    // Set data
    //float dens_221_f[221] = {1.016692e-33,2.333626e-34,8.367814e-34,6.744978e-35,4.493371e-36,4.396106e-37,1.908333e-34,4.848228e-35,7.228556e-34,1.717567e-38,6.815741e-37,2.776831e-36,1.143339e-34,3.467837e-35,2.173183e-36,2.919116e-35,3.249745e-13,1.280894e-13,2.872959e-13,5.841356e-14,1.052731e-14,4.778827e-14,2.419135e-15,1.127769e-13,4.739891e-14,2.618893e-13,3.192557e-15,6.648584e-15,4.077459e-07,2.149901e-07,3.746791e-07,1.251948e-07,3.841547e-08,1.084211e-07,3.515827e-07,1.679403e-08,2.794462e-08,3.479711e-05,2.007635e-05,1.120590e-05,2.301993e-06,3.572237e-06,1.566220e-05,5.111461e-31,5.111677e-31,5.111461e-31,5.111597e-31,5.111461e-31,5.111840e-31,5.111245e-31,5.111542e-31,5.111677e-31,5.111786e-31,5.111677e-31,5.111892e-31,5.111867e-31,5.111594e-31,5.111542e-31,5.111461e-31,5.111461e-31,5.111489e-31,5.111542e-31,5.111299e-31,5.111867e-31,5.111786e-31,5.111299e-31,1.202872e-12,1.203062e-12,1.203132e-12,1.203147e-12,1.203080e-12,1.203036e-12,1.203051e-12,1.202950e-12,1.203056e-12,1.203025e-12,1.203067e-12,1.201019e-12,1.202777e-12,1.202918e-12,1.198146e-12,1.202207e-12,1.202323e-12,1.202721e-12,1.202982e-12,1.201824e-12,1.203016e-12,1.203111e-12,1.203109e-12,1.203064e-12,1.202214e-12,1.203094e-12,1.056859e-07,1.069813e-07,1.068918e-07,1.069731e-07,1.067879e-07,9.925203e-08,1.065169e-07,1.067720e-07,9.567351e-08,1.061269e-07,1.068738e-07,1.002584e-07,1.067610e-07,1.069669e-07,1.020017e-07,1.054180e-07,1.061756e-07,9.844285e-08,1.040931e-07,1.043744e-07,1.052831e-07,1.062094e-07,1.033102e-07,1.064327e-07,1.068430e-07,1.067436e-07,1.064088e-07,1.040096e-07,1.067188e-07,5.449067e-06,2.706551e-06,5.836964e-06,3.695634e-06,5.801161e-06,5.833626e-06,3.451926e-06,5.762271e-06,4.397482e-06,5.673675e-06,5.758306e-06,3.997644e-06,5.560026e-06,5.793905e-06,4.525559e-06,5.752838e-06,5.831018e-06,5.168859e-06,5.356980e-06,5.582546e-06,4.978430e-06,5.640204e-06,5.102124e-06,2.601504e-05,9.496978e-06,3.058765e-05,1.358994e-05,3.008526e-05,2.710820e-05,3.054102e-05,1.249342e-05,2.956473e-05,1.764504e-05,2.846226e-05,2.951743e-05,1.532756e-05,2.717309e-05,1.355043e-05,2.998640e-05,1.852374e-05,2.944374e-05,1.020576e-05,3.050481e-05,2.341930e-05,2.510869e-05,2.741899e-05,2.183935e-05,2.806830e-05,2.283266e-05,5.046140e-05,1.499352e-05,6.507664e-05,2.169488e-05,6.326965e-05,4.041267e-05,6.489926e-05,1.996210e-05,6.149049e-05,3.179689e-05,6.107018e-05,4.349307e-05,4.785749e-05,5.461262e-05,3.951244e-05,5.657595e-05,4.188267e-05,6.643681e-05,1.701268e-05,9.193084e-05,2.527302e-05,8.849833e-05,3.780743e-05,9.155914e-05,2.338526e-05,8.530516e-05,3.873166e-05,8.451808e-05,7.331888e-05,4.973260e-05,7.654892e-05,5.304757e-05,7.487628e-05,1.748799e-05,1.078457e-04,4.236045e-05,9.790303e-05,4.905088e-17,3.745068e-17,4.379298e-19,1.966976e-19,5.755041e-15,2.728153e-14,1.389058e-16,3.278741e-16,4.960657e-15,6.570254e-15,9.327300e-19,1.487219e-05,5.521405e-06,1.366262e-05};
    //float contracted_grad_221_f[221] = {0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,2.631121e-24,4.275903e-25,2.068908e-24,9.222840e-26,3.229578e-27,6.229285e-26,1.811449e-28,3.334638e-25,6.130454e-26,1.726994e-24,3.120054e-28,1.313191e-27,1.252649e-12,3.835113e-13,1.071992e-12,1.400556e-13,1.523680e-14,1.070453e-13,9.533463e-13,3.176699e-15,8.346576e-15,2.204637e-09,8.742831e-10,3.206804e-10,2.002568e-11,4.366787e-11,3.086138e-10,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,0.000000e+00,3.370783e-23,3.372360e-23,3.372715e-23,3.372821e-23,3.372406e-23,3.372065e-23,3.372238e-23,3.371434e-23,3.372298e-23,3.372095e-23,3.372386e-23,3.357765e-23,3.370100e-23,3.371264e-23,3.338174e-23,3.366109e-23,3.366948e-23,3.369719e-23,3.371641e-23,3.363387e-23,3.371924e-23,3.372586e-23,3.372542e-23,3.372177e-23,3.366073e-23,3.372460e-23,1.019537e-13,1.054808e-13,1.052235e-13,1.054570e-13,1.049289e-13,8.677356e-14,1.041766e-13,1.048868e-13,7.931585e-14,1.031177e-13,1.051718e-13,8.897470e-14,1.048535e-13,1.054386e-13,9.293591e-14,1.012597e-13,1.032553e-13,8.502833e-14,9.791433e-14,9.860958e-14,1.009087e-13,1.033391e-13,9.601215e-14,1.039444e-13,1.050876e-13,1.048057e-13,1.038799e-13,9.771355e-14,1.047368e-13,1.162748e-10,2.334933e-11,1.424700e-10,4.534937e-11,1.397742e-10,1.422207e-10,3.933528e-11,1.369135e-10,6.691021e-11,1.306937e-10,1.366455e-10,5.385416e-11,1.231582e-10,1.392328e-10,7.169444e-11,1.362370e-10,1.420250e-10,1.005558e-10,1.108657e-10,1.246111e-10,9.106069e-11,1.284195e-10,9.712042e-11,1.212137e-09,1.356080e-10,1.951985e-09,3.867117e-10,1.850601e-09,8.758271e-09,1.942269e-09,3.268466e-10,1.751971e-09,5.571538e-10,1.559754e-09,1.742656e-09,4.531791e-10,1.365493e-09,1.377643e-09,1.831493e-09,6.010568e-10,1.729588e-09,4.316582e-10,1.934755e-09,9.375115e-10,1.104771e-09,1.400412e-09,8.067573e-10,1.497223e-09,8.842579e-10,2.398833e-09,2.080791e-10,4.198249e-09,1.379833e-09,3.853848e-09,2.323771e-08,4.159398e-09,1.061023e-09,3.564113e-09,1.848764e-09,3.496477e-09,2.089923e-09,2.246474e-09,2.720921e-09,1.991728e-09,2.908715e-09,2.046485e-09,3.957297e-09,4.837907e-10,3.625368e-09,3.493492e-09,3.473267e-09,2.693226e-08,3.592708e-09,2.585125e-09,3.433550e-09,4.815363e-09,3.420134e-09,3.636069e-09,4.738457e-09,3.512618e-09,4.680167e-09,7.756806e-09,1.712467e-09,2.265633e-09,9.828216e-09,3.841888e-09,8.604033e-32,5.062453e-32,7.974179e-36,1.646879e-36,9.898778e-28,2.081598e-26,6.652570e-31,3.592657e-30,7.397754e-28,1.283103e-27,3.536233e-35,6.664909e-10,7.040826e-11,4.714138e-10};
    //G2G::vec_type<float,4> grad_221_f[221] = {G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000001,-0.000001,0.000001,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000001,0),G2G::vec_type<float,4>(0.000000,-0.000001,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000001,0.000001,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000035,-0.000017,0.000026,0),G2G::vec_type<float,4>(0.000009,0.000001,0.000028,0),G2G::vec_type<float,4>(0.000013,0.000007,0.000010,0),G2G::vec_type<float,4>(0.000003,0.000003,0.000001,0),G2G::vec_type<float,4>(0.000002,0.000004,0.000005,0),G2G::vec_type<float,4>(-0.000000,0.000017,-0.000003,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000008,0.000000,0.000008,0),G2G::vec_type<float,4>(0.000002,0.000004,0.000002,0),G2G::vec_type<float,4>(0.000007,-0.000007,0.000007,0),G2G::vec_type<float,4>(0.000002,0.000005,0.000004,0),G2G::vec_type<float,4>(0.000005,-0.000005,0.000009,0),G2G::vec_type<float,4>(0.000005,-0.000009,0.000005,0),G2G::vec_type<float,4>(0.000004,0.000004,0.000002,0),G2G::vec_type<float,4>(0.000009,-0.000005,0.000005,0),G2G::vec_type<float,4>(0.000002,0.000004,0.000007,0),G2G::vec_type<float,4>(0.000003,-0.000003,0.000011,0),G2G::vec_type<float,4>(0.000003,-0.000011,0.000003,0),G2G::vec_type<float,4>(0.000006,0.000004,0.000001,0),G2G::vec_type<float,4>(0.000010,-0.000003,0.000003,0),G2G::vec_type<float,4>(0.000008,-0.000008,0.000004,0),G2G::vec_type<float,4>(0.000005,0.000004,0.000005,0),G2G::vec_type<float,4>(0.000008,-0.000004,0.000008,0),G2G::vec_type<float,4>(0.000004,-0.000008,0.000008,0),G2G::vec_type<float,4>(0.000010,0.000001,0.000003,0),G2G::vec_type<float,4>(0.000003,0.000001,0.000010,0),G2G::vec_type<float,4>(0.000009,-0.000001,0.000006,0),G2G::vec_type<float,4>(0.000008,0.000002,0.000005,0),G2G::vec_type<float,4>(0.000006,-0.000002,0.000010,0),G2G::vec_type<float,4>(0.000005,0.000002,0.000008,0),G2G::vec_type<float,4>(0.000025,0.000007,0.000024,0),G2G::vec_type<float,4>(0.000003,0.000011,0.000002,0),G2G::vec_type<float,4>(0.000026,-0.000025,0.000025,0),G2G::vec_type<float,4>(0.000001,0.000019,0.000006,0),G2G::vec_type<float,4>(0.000019,-0.000018,0.000034,0),G2G::vec_type<float,4>(0.000055,-0.000034,0.000067,0),G2G::vec_type<float,4>(0.000019,-0.000035,0.000019,0),G2G::vec_type<float,4>(0.000007,0.000017,-0.000000,0),G2G::vec_type<float,4>(0.000034,-0.000017,0.000018,0),G2G::vec_type<float,4>(0.000001,0.000018,0.000015,0),G2G::vec_type<float,4>(0.000010,-0.000008,0.000037,0),G2G::vec_type<float,4>(0.000011,-0.000039,0.000010,0),G2G::vec_type<float,4>(0.000013,0.000017,-0.000002,0),G2G::vec_type<float,4>(0.000036,-0.000007,0.000008,0),G2G::vec_type<float,4>(0.000025,-0.000013,0.000025,0),G2G::vec_type<float,4>(0.000030,-0.000028,0.000012,0),G2G::vec_type<float,4>(0.000012,0.000019,0.000010,0),G2G::vec_type<float,4>(0.000029,-0.000010,0.000028,0),G2G::vec_type<float,4>(0.000011,-0.000005,0.000017,0),G2G::vec_type<float,4>(0.000013,-0.000029,0.000030,0),G2G::vec_type<float,4>(0.000029,0.000008,0.000006,0),G2G::vec_type<float,4>(0.000009,0.000007,0.000031,0),G2G::vec_type<float,4>(0.000033,-0.000001,0.000018,0),G2G::vec_type<float,4>(0.000022,0.000014,0.000010,0),G2G::vec_type<float,4>(0.000020,-0.000002,0.000033,0),G2G::vec_type<float,4>(0.000013,0.000014,0.000023,0),G2G::vec_type<float,4>(0.000031,0.000025,0.000028,0),G2G::vec_type<float,4>(-0.000001,0.000014,-0.000001,0),G2G::vec_type<float,4>(0.000040,-0.000035,0.000037,0),G2G::vec_type<float,4>(-0.000007,0.000036,-0.000002,0),G2G::vec_type<float,4>(0.000026,-0.000020,0.000053,0),G2G::vec_type<float,4>(0.000077,-0.000088,0.000098,0),G2G::vec_type<float,4>(0.000028,-0.000053,0.000025,0),G2G::vec_type<float,4>(0.000001,0.000031,-0.000009,0),G2G::vec_type<float,4>(0.000053,-0.000018,0.000021,0),G2G::vec_type<float,4>(0.000007,0.000042,0.000003,0),G2G::vec_type<float,4>(0.000043,-0.000007,0.000040,0),G2G::vec_type<float,4>(0.000036,0.000028,-0.000003,0),G2G::vec_type<float,4>(0.000004,0.000027,0.000039,0),G2G::vec_type<float,4>(0.000047,0.000011,0.000020,0),G2G::vec_type<float,4>(0.000024,0.000037,0.000004,0),G2G::vec_type<float,4>(0.000025,0.000010,0.000046,0),G2G::vec_type<float,4>(0.000010,0.000037,0.000023,0),G2G::vec_type<float,4>(0.000027,0.000054,0.000019,0),G2G::vec_type<float,4>(-0.000008,0.000019,-0.000007,0),G2G::vec_type<float,4>(0.000041,-0.000028,0.000034,0),G2G::vec_type<float,4>(-0.000018,0.000055,-0.000013,0),G2G::vec_type<float,4>(0.000021,-0.000008,0.000054,0),G2G::vec_type<float,4>(0.000066,-0.000120,0.000090,0),G2G::vec_type<float,4>(0.000024,-0.000052,0.000017,0),G2G::vec_type<float,4>(-0.000008,0.000046,-0.000019,0),G2G::vec_type<float,4>(0.000057,-0.000004,0.000011,0),G2G::vec_type<float,4>(-0.000003,0.000069,-0.000010,0),G2G::vec_type<float,4>(0.000044,0.000011,0.000037,0),G2G::vec_type<float,4>(0.000048,0.000036,0.000009,0),G2G::vec_type<float,4>(0.000018,0.000066,-0.000011,0),G2G::vec_type<float,4>(0.000019,0.000034,0.000044,0),G2G::vec_type<float,4>(-0.000001,0.000067,0.000014,0),G2G::vec_type<float,4>(0.000023,0.000084,0.000011,0),G2G::vec_type<float,4>(-0.000016,0.000035,-0.000016,0),G2G::vec_type<float,4>(0.000037,-0.000014,0.000026,0),G2G::vec_type<float,4>(-0.000009,0.000097,-0.000018,0),G2G::vec_type<float,4>(0.000042,0.000035,0.000030,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000009,-0.000018,0.000016,0),G2G::vec_type<float,4>(0.000005,0.000007,0.0000010,0),G2G::vec_type<float,4>(0.000011,0.000005,0.000018,0)};
    //G2G::vec_type<float,4> hess1_221_f[221] = {G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000001,0.000001,0.000001,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000001,0),G2G::vec_type<float,4>(0.000000,0.000002,0.000000,0),G2G::vec_type<float,4>(0.000001,-0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000001,0.000001,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000014,-0.000019,-0.000005,0),G2G::vec_type<float,4>(-0.000011,-0.000016,0.000027,0),G2G::vec_type<float,4>(0.000007,-0.000005,0.000001,0),G2G::vec_type<float,4>(0.000003,0.000002,-0.000002,0),G2G::vec_type<float,4>(-0.000002,0.000002,0.000003,0),G2G::vec_type<float,4>(-0.000004,0.000095,0.000018,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000001,0),G2G::vec_type<float,4>(0.000000,0.000001,0.000000,0),G2G::vec_type<float,4>(0.000001,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000001,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000001,0),G2G::vec_type<float,4>(0.000000,0.000001,0.000000,0),G2G::vec_type<float,4>(0.000001,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000001,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,-0.000000,0.000001,0),G2G::vec_type<float,4>(-0.000000,-0.000000,0.000001,0),G2G::vec_type<float,4>(-0.000000,0.000001,-0.000000,0),G2G::vec_type<float,4>(0.000001,-0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000001,-0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000001,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000001,0),G2G::vec_type<float,4>(0.000001,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000001,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000001,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000001,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,0.000001,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000001,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000001,0),G2G::vec_type<float,4>(0.000000,0.000001,-0.000000,0),G2G::vec_type<float,4>(0.000007,-0.000004,0.000007,0),G2G::vec_type<float,4>(-0.000000,0.000006,-0.000000,0),G2G::vec_type<float,4>(0.000005,0.000004,0.000004,0),G2G::vec_type<float,4>(-0.000002,0.000003,0.000002,0),G2G::vec_type<float,4>(0.000001,0.000001,0.000011,0),G2G::vec_type<float,4>(0.000001,0.000011,0.000001,0),G2G::vec_type<float,4>(0.000002,0.000003,-0.000002,0),G2G::vec_type<float,4>(0.000011,0.000001,0.000001,0),G2G::vec_type<float,4>(-0.000002,-0.000000,0.000008,0),G2G::vec_type<float,4>(-0.000002,-0.000002,0.000016,0),G2G::vec_type<float,4>(-0.000002,0.000016,-0.000002,0),G2G::vec_type<float,4>(0.000007,0.000000,-0.000003,0),G2G::vec_type<float,4>(0.000016,-0.000002,-0.000002,0),G2G::vec_type<float,4>(0.000007,0.000007,-0.000001,0),G2G::vec_type<float,4>(0.000003,0.000000,0.000002,0),G2G::vec_type<float,4>(0.000007,-0.000002,0.000007,0),G2G::vec_type<float,4>(-0.000001,0.000007,0.000007,0),G2G::vec_type<float,4>(0.000014,-0.000004,-0.000002,0),G2G::vec_type<float,4>(-0.000001,-0.000004,0.000015,0),G2G::vec_type<float,4>(0.000013,-0.000003,0.000002,0),G2G::vec_type<float,4>(0.000009,-0.000002,0.000001,0),G2G::vec_type<float,4>(0.000003,-0.000003,0.000013,0),G2G::vec_type<float,4>(0.000001,-0.000003,0.000009,0),G2G::vec_type<float,4>(0.000006,-0.000021,0.000003,0),G2G::vec_type<float,4>(-0.000001,0.000033,0.000004,0),G2G::vec_type<float,4>(0.000003,-0.000001,0.000001,0),G2G::vec_type<float,4>(-0.000012,0.000018,-0.000012,0),G2G::vec_type<float,4>(-0.000009,-0.000012,0.000021,0),G2G::vec_type<float,4>(0.000108,0.000012,0.000166,0),G2G::vec_type<float,4>(-0.000008,0.000020,-0.000010,0),G2G::vec_type<float,4>(-0.000008,0.000020,-0.000010,0),G2G::vec_type<float,4>(0.000022,-0.000013,-0.000011,0),G2G::vec_type<float,4>(-0.000016,0.000001,-0.000004,0),G2G::vec_type<float,4>(-0.000017,-0.000021,0.000032,0),G2G::vec_type<float,4>(-0.000017,0.000034,-0.000019,0),G2G::vec_type<float,4>(-0.000002,0.000006,-0.000014,0),G2G::vec_type<float,4>(0.000031,-0.000021,-0.000019,0),G2G::vec_type<float,4>(0.000058,0.000034,0.000070,0),G2G::vec_type<float,4>(0.000011,0.000006,-0.000017,0),G2G::vec_type<float,4>(-0.000008,0.000002,-0.000012,0),G2G::vec_type<float,4>(0.000010,-0.000019,0.000007,0),G2G::vec_type<float,4>(0.000028,0.000035,0.000056,0),G2G::vec_type<float,4>(-0.000016,0.000008,0.000010,0),G2G::vec_type<float,4>(0.000021,-0.000019,-0.000019,0),G2G::vec_type<float,4>(-0.000016,-0.000021,0.000022,0),G2G::vec_type<float,4>(0.000023,-0.000023,-0.000008,0),G2G::vec_type<float,4>(0.000007,-0.000011,-0.000014,0),G2G::vec_type<float,4>(-0.000005,-0.000023,0.000021,0),G2G::vec_type<float,4>(-0.000010,-0.000013,0.000005,0),G2G::vec_type<float,4>(-0.000018,-0.000049,-0.000029,0),G2G::vec_type<float,4>(0.000003,0.000100,0.000020,0),G2G::vec_type<float,4>(-0.000023,-0.000037,-0.000030,0),G2G::vec_type<float,4>(-0.000023,0.000058,-0.000029,0),G2G::vec_type<float,4>(-0.000041,-0.000054,0.000001,0),G2G::vec_type<float,4>(0.000149,0.000116,0.000241,0),G2G::vec_type<float,4>(-0.000040,-0.000004,-0.000046,0),G2G::vec_type<float,4>(-0.000022,0.000066,-0.000011,0),G2G::vec_type<float,4>(0.000008,-0.000054,-0.000048,0),G2G::vec_type<float,4>(-0.000032,0.000016,-0.000039,0),G2G::vec_type<float,4>(-0.000012,-0.000062,-0.000021,0),G2G::vec_type<float,4>(0.000003,-0.000041,-0.000050,0),G2G::vec_type<float,4>(-0.000046,-0.000047,-0.000002,0),G2G::vec_type<float,4>(0.000007,-0.000061,-0.000044,0),G2G::vec_type<float,4>(-0.000016,-0.000018,-0.000045,0),G2G::vec_type<float,4>(-0.000035,-0.000062,-0.000001,0),G2G::vec_type<float,4>(-0.000038,-0.000023,-0.000025,0),G2G::vec_type<float,4>(-0.000040,-0.000078,-0.000057,0),G2G::vec_type<float,4>(0.000006,0.000207,0.000035,0),G2G::vec_type<float,4>(-0.000058,-0.000088,-0.000072,0),G2G::vec_type<float,4>(-0.000028,0.000128,-0.000030,0),G2G::vec_type<float,4>(-0.000083,-0.000109,-0.000022,0),G2G::vec_type<float,4>(0.000146,0.000282,0.000245,0),G2G::vec_type<float,4>(-0.000084,-0.000038,-0.000096,0),G2G::vec_type<float,4>(-0.000027,0.000142,-0.000004,0),G2G::vec_type<float,4>(-0.000007,-0.000109,-0.000095,0),G2G::vec_type<float,4>(-0.000047,0.000050,-0.000053,0),G2G::vec_type<float,4>(-0.000038,-0.000118,-0.000055,0),G2G::vec_type<float,4>(-0.000003,-0.000105,-0.000083,0),G2G::vec_type<float,4>(-0.000026,-0.000015,-0.000070,0),G2G::vec_type<float,4>(-0.000070,-0.000109,-0.000020,0),G2G::vec_type<float,4>(-0.000064,-0.000024,-0.000042,0),G2G::vec_type<float,4>(-0.000057,-0.000111,-0.000077,0),G2G::vec_type<float,4>(-0.000002,0.000337,0.000033,0),G2G::vec_type<float,4>(-0.000100,-0.000144,-0.000121,0),G2G::vec_type<float,4>(-0.000057,0.000094,-0.000062,0),G2G::vec_type<float,4>(-0.000065,-0.000183,-0.000088,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,-0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000004,0.000013,0.000007,0),G2G::vec_type<float,4>(0.000001,0.000011,-0.000001,0),G2G::vec_type<float,4>(-0.000000,-0.000009,0.000014,0)};
    //G2G::vec_type<float,4> hess2_221_f[221] = {G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000001,0.000001,-0.000001,0),G2G::vec_type<float,4>(-0.000000,0.000001,-0.000000,0),G2G::vec_type<float,4>(-0.000001,0.000000,-0.000001,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000001,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000017,0.000027,-0.000011,0),G2G::vec_type<float,4>(0.000003,0.000011,0.000005,0),G2G::vec_type<float,4>(0.000010,0.000011,0.000009,0),G2G::vec_type<float,4>(0.000005,0.000000,0.000001,0),G2G::vec_type<float,4>(0.000003,0.000002,0.000007,0),G2G::vec_type<float,4>(-0.000028,0.000011,-0.000050,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000001,0.000010,0.000001,0),G2G::vec_type<float,4>(0.000002,0.000001,0.000002,0),G2G::vec_type<float,4>(-0.000008,0.000008,-0.000008,0),G2G::vec_type<float,4>(0.000003,0.000002,0.000006,0),G2G::vec_type<float,4>(-0.000005,0.000008,-0.000008,0),G2G::vec_type<float,4>(-0.000008,0.000005,-0.000008,0),G2G::vec_type<float,4>(0.000006,0.000001,0.000003,0),G2G::vec_type<float,4>(-0.000008,0.000008,-0.000004,0),G2G::vec_type<float,4>(0.000002,0.000002,0.000007,0),G2G::vec_type<float,4>(-0.000002,0.000006,-0.000006,0),G2G::vec_type<float,4>(-0.000006,0.000002,-0.000006,0),G2G::vec_type<float,4>(0.000007,0.000001,0.000002,0),G2G::vec_type<float,4>(-0.000005,0.000005,-0.000001,0),G2G::vec_type<float,4>(-0.000011,0.000005,-0.000005,0),G2G::vec_type<float,4>(0.000005,0.000005,0.000005,0),G2G::vec_type<float,4>(-0.000005,0.000011,-0.000005,0),G2G::vec_type<float,4>(-0.000005,0.000005,-0.000011,0),G2G::vec_type<float,4>(0.000002,0.000005,0.000001,0),G2G::vec_type<float,4>(0.000001,0.000006,0.000002,0),G2G::vec_type<float,4>(-0.000002,0.000010,-0.000001,0),G2G::vec_type<float,4>(0.000005,0.000007,0.000003,0),G2G::vec_type<float,4>(-0.000001,0.000010,-0.000002,0),G2G::vec_type<float,4>(0.000003,0.000008,0.000004,0),G2G::vec_type<float,4>(0.000011,0.000022,0.000012,0),G2G::vec_type<float,4>(-0.000008,0.000006,-0.000012,0),G2G::vec_type<float,4>(-0.000021,0.000022,-0.000020,0),G2G::vec_type<float,4>(0.000007,-0.000005,0.000015,0),G2G::vec_type<float,4>(-0.000010,0.000022,-0.000020,0),G2G::vec_type<float,4>(-0.000080,0.000147,-0.000094,0),G2G::vec_type<float,4>(-0.000022,0.000011,-0.000021,0),G2G::vec_type<float,4>(0.000013,-0.000005,0.000001,0),G2G::vec_type<float,4>(-0.000019,0.000020,-0.000008,0),G2G::vec_type<float,4>(0.000007,-0.000006,0.000026,0),G2G::vec_type<float,4>(-0.000000,0.000012,-0.000008,0),G2G::vec_type<float,4>(-0.000013,0.000002,-0.000012,0),G2G::vec_type<float,4>(0.000022,-0.000008,0.000003,0),G2G::vec_type<float,4>(-0.000006,0.000008,0.000002,0),G2G::vec_type<float,4>(-0.000056,0.000071,-0.000057,0),G2G::vec_type<float,4>(-0.000029,0.000011,-0.000009,0),G2G::vec_type<float,4>(0.000020,0.000001,0.000020,0),G2G::vec_type<float,4>(-0.000009,0.000028,-0.000008,0),G2G::vec_type<float,4>(-0.000036,0.000043,-0.000048,0),G2G::vec_type<float,4>(-0.000012,0.000012,-0.000029,0),G2G::vec_type<float,4>(0.000016,0.000004,0.000008,0),G2G::vec_type<float,4>(0.000007,0.000008,0.000015,0),G2G::vec_type<float,4>(0.000002,0.000021,0.000003,0),G2G::vec_type<float,4>(0.000022,0.000007,0.000014,0),G2G::vec_type<float,4>(0.000002,0.000023,0.000002,0),G2G::vec_type<float,4>(0.000014,0.000010,0.000022,0),G2G::vec_type<float,4>(0.000034,0.000015,0.000036,0),G2G::vec_type<float,4>(-0.000039,0.000016,-0.000054,0),G2G::vec_type<float,4>(-0.000021,0.000024,-0.000017,0),G2G::vec_type<float,4>(-0.000001,-0.000014,0.000013,0),G2G::vec_type<float,4>(-0.000003,0.000022,-0.000013,0),G2G::vec_type<float,4>(-0.000189,0.000199,-0.000226,0),G2G::vec_type<float,4>(-0.000022,0.000007,-0.000018,0),G2G::vec_type<float,4>(0.000009,-0.000009,-0.000019,0),G2G::vec_type<float,4>(-0.000014,0.000018,0.000003,0),G2G::vec_type<float,4>(0.000035,-0.000013,0.000032,0),G2G::vec_type<float,4>(0.000002,0.000031,0.000005,0),G2G::vec_type<float,4>(0.000045,-0.000013,0.000021,0),G2G::vec_type<float,4>(0.000021,-0.000005,0.000046,0),G2G::vec_type<float,4>(0.000022,0.000016,0.000021,0),G2G::vec_type<float,4>(0.000049,-0.000008,0.000029,0),G2G::vec_type<float,4>(0.000017,0.000021,0.000023,0),G2G::vec_type<float,4>(0.000032,-0.000003,0.000050,0),G2G::vec_type<float,4>(0.000060,0.000018,0.000061,0),G2G::vec_type<float,4>(-0.000067,0.000023,-0.000096,0),G2G::vec_type<float,4>(-0.000019,0.000026,-0.000011,0),G2G::vec_type<float,4>(-0.000014,-0.000012,0.000008,0),G2G::vec_type<float,4>(0.000008,0.000023,-0.000004,0),G2G::vec_type<float,4>(-0.000262,0.000177,-0.000321,0),G2G::vec_type<float,4>(-0.000021,0.000001,-0.000012,0),G2G::vec_type<float,4>(0.000009,-0.000006,-0.000047,0),G2G::vec_type<float,4>(-0.000007,0.000016,0.000017,0),G2G::vec_type<float,4>(0.000050,-0.000013,0.000041,0),G2G::vec_type<float,4>(0.000016,0.000038,0.000021,0),G2G::vec_type<float,4>(0.000046,0.000016,0.000040,0),G2G::vec_type<float,4>(0.000077,-0.000010,0.000040,0),G2G::vec_type<float,4>(0.000036,0.000023,0.000049,0),G2G::vec_type<float,4>(0.000048,-0.000006,0.000079,0),G2G::vec_type<float,4>(0.000085,0.000042,0.000083,0),G2G::vec_type<float,4>(-0.000070,0.000022,-0.000113,0),G2G::vec_type<float,4>(-0.000028,0.000039,-0.000017,0),G2G::vec_type<float,4>(0.000072,0.000003,0.000055,0),G2G::vec_type<float,4>(0.000023,0.000061,0.000029,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(0.000000,0.000000,0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000000,0.000000,-0.000000,0),G2G::vec_type<float,4>(-0.000011,0.000010,-0.000019,0),G2G::vec_type<float,4>(0.000004,0.000001,-0.000003,0),G2G::vec_type<float,4>(0.000006,0.000014,0.000009,0)};

    float* partial_densities_cpu = dens_221_f;
    float* point_weights_cpu = dens_221_f;

    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemcpy(point_weights_gpu_in, point_weights_cpu, size, hipMemcpyHostToDevice);
    hipMemcpy(partial_density_gpu_in, partial_densities_cpu, size, hipMemcpyHostToDevice);

    hipMemcpy(dxyz_gpu_in, grad_221_f, vec_size, hipMemcpyHostToDevice);
    hipMemcpy(dd1_gpu_in, hess1_221_f, vec_size, hipMemcpyHostToDevice);
    hipMemcpy(dd2_gpu_in, hess2_221_f, vec_size, hipMemcpyHostToDevice);

    hipMemset(dxyz_gpu_accum,0,vec_size);
    hipMemset(dd1_gpu_accum,0,vec_size);
    hipMemset(dd2_gpu_accum,0,vec_size);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<float,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    /////////////////////////////////
    // LIBXC VERSION
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    // ACCUMULATE DATA
    gpu_accumulate_point_for_libxc<float,true,true,false><<<blocksPerGrid,threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in, dd1_gpu_in, dd2_gpu_in,
	partial_density_gpu_accum, dxyz_gpu_accum, dd1_gpu_accum, dd2_gpu_accum);

    // EXCHANGE_CORRELATION WITH LIBXC_CUDA
    // Calculate exc_corr and y2a
    libxc_exchange_correlation_gpu<float, true, true, false> (&libxcProxy,
	energy_gpu_in,
	factor_gpu_in,
	number_of_points,
	partial_density_gpu_accum,
	dxyz_gpu_accum,
        dd1_gpu_accum,
	dd2_gpu_accum);
    
    // MERGE RESULTS.
    // Join the results.
    gpu_accumulate_energy_and_forces_from_libxc<float, true, true, false><<<blocksPerGrid, threadsPerBlock>>> (
	energy_gpu_in,
	factor_gpu_in,
	point_weights_gpu_in,
	number_of_points,
	partial_density_gpu_accum);

    ///////////////////////////
    // Print libxc results
    // Allocate the host input vectors
    float *energy_cpu = (float *)malloc(size);
    float *factor_cpu = (float *)malloc(size);

    // Copy the vectors from gpu to cpu
    // Be aware that energy_gpu can be NULL.
    err = hipMemcpy(energy_cpu, energy_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector energy_gpu_in from device to host!\n");
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(factor_cpu, factor_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector factor_gpu_in from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (NULL, NULL, NULL, energy_cpu,
	factor_cpu, NULL, NULL, number_of_points);

    ////////////////////////////
    // Free Memory CPU
    free(energy_cpu);
    free(factor_cpu);

    ///////////////////////////
    // Free memory GPU
    hipFree (point_weights_gpu_in);
    hipFree (partial_density_gpu_in);
    hipFree (partial_density_gpu_accum);
    hipFree (energy_gpu_in);
    hipFree (factor_gpu_in);
    hipFree (dxyz_gpu_in);
    hipFree (dd1_gpu_in);
    hipFree (dd2_gpu_in);
    hipFree (dxyz_gpu_accum);
    hipFree (dd1_gpu_accum);
    hipFree (dd2_gpu_accum);
}


//////////////////////////////////////////////////////////////////
// Test: iteration_test0010()
//
// Simulamos una iteracion del core de lio
// para el caso el tipo de datos Float y
// el template con los valores
// <false, true, false>
//
void iteration_test0010() {
    printf("accumulate_data_for_libxc_test0010() FLOATS\n");
    hipError_t err = hipSuccess;

    uint number_of_points = 221;
    uint vec_size = sizeof(G2G::vec_type<float,4>)*number_of_points;

    // Input
    G2G::vec_type<float,4>* dxyz_gpu_in = NULL;
    G2G::vec_type<float,4>* dd1_gpu_in = NULL;
    G2G::vec_type<float,4>* dd2_gpu_in = NULL;

    // Accumulate
    G2G::vec_type<float,4>* dxyz_gpu_accum = NULL;
    G2G::vec_type<float,4>* dd1_gpu_accum = NULL;
    G2G::vec_type<float,4>* dd2_gpu_accum = NULL;

    // Alloc memory for the arrays.
    hipMalloc ((void**)&dxyz_gpu_in, vec_size);
    hipMalloc ((void**)&dd1_gpu_in, vec_size);
    hipMalloc ((void**)&dd2_gpu_in, vec_size);

    hipMalloc ((void**)&dxyz_gpu_accum, vec_size);
    hipMalloc ((void**)&dd1_gpu_accum, vec_size);
    hipMalloc ((void**)&dd2_gpu_accum, vec_size);

    // Now the arrays for energy, factors, point_weight and partial_density
    float *point_weights_gpu_in = NULL;
    float *partial_density_gpu_in = NULL;
    float *partial_density_gpu_accum = NULL;

    // Create the arrays in CUDA memory.
    uint size = number_of_points * sizeof(float);
    err = hipMalloc((void**)&point_weights_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector point_weights_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_in!\n");
    }

    err = hipMalloc((void**)&partial_density_gpu_accum, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector partial_density_gpu_accum!\n");
    }

    // Now the arrays for energy, factors
    float *energy_gpu_in = NULL;
    float *factor_gpu_in = NULL;
    err = hipMalloc((void**)&energy_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector energy_gpu!\n");
    }

    err = hipMalloc((void**)&factor_gpu_in, size);
    if (err != hipSuccess)
    {
	printf("Failed to allocate vector factor_gpu!\n");
    }

    // Launch the CUDA Kernel
    int numElements = 221;
    int threadsPerBlock = 32;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    uint block_height = 1;

    ///////////////////////
    // Set data
    float* partial_densities_cpu = dens_221_f;
    float* point_weights_cpu = dens_221_f;

    hipMemset(energy_gpu_in, 0, size);
    hipMemset(factor_gpu_in, 0, size);
    hipMemcpy(point_weights_gpu_in, point_weights_cpu, size, hipMemcpyHostToDevice);
    hipMemcpy(partial_density_gpu_in, partial_densities_cpu, size, hipMemcpyHostToDevice);

    hipMemcpy(dxyz_gpu_in, grad_221_f, vec_size, hipMemcpyHostToDevice);
    hipMemcpy(dd1_gpu_in, hess1_221_f, vec_size, hipMemcpyHostToDevice);
    hipMemcpy(dd2_gpu_in, hess2_221_f, vec_size, hipMemcpyHostToDevice);

    hipMemset(dxyz_gpu_accum,0,vec_size);
    hipMemset(dd1_gpu_accum,0,vec_size);
    hipMemset(dd2_gpu_accum,0,vec_size);

    // Create the libxcproxy
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<float,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    /////////////////////////////////
    // LIBXC VERSION
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    // ACCUMULATE DATA
    gpu_accumulate_point_for_libxc<float,false,true,false><<<blocksPerGrid,threadsPerBlock>>> (point_weights_gpu_in,
	number_of_points, block_height,
	partial_density_gpu_in, dxyz_gpu_in, dd1_gpu_in, dd2_gpu_in,
	partial_density_gpu_accum, dxyz_gpu_accum, dd1_gpu_accum, dd2_gpu_accum);

    // EXCHANGE_CORRELATION WITH LIBXC_CUDA
    // Calculate exc_corr and y2a
    libxc_exchange_correlation_gpu<float, false, true, false> (&libxcProxy,
	NULL,
	factor_gpu_in,
	number_of_points,
	partial_density_gpu_accum,
	dxyz_gpu_accum,
        dd1_gpu_accum,
	dd2_gpu_accum);

    // MERGE RESULTS.
    // Join the results.
    gpu_accumulate_energy_and_forces_from_libxc<float, false, true, false><<<blocksPerGrid, threadsPerBlock>>> (
	NULL,
	factor_gpu_in,
	point_weights_gpu_in,
	number_of_points,
	partial_density_gpu_accum);

    ///////////////////////////
    // Print libxc results
    // Allocate the host input vectors
    float *energy_cpu = (float *)malloc(size);
    float *factor_cpu = (float *)malloc(size);

    // Copy the vectors from gpu to cpu
    // Be aware that energy_gpu can be NULL.
    //err = hipMemcpy(energy_cpu, energy_gpu_in, size, hipMemcpyDeviceToHost);
    //if (err != hipSuccess)
    //{
    //    printf("Failed to copy vector energy_gpu_in from device to host!\n");
    //    exit(EXIT_FAILURE);
    //}

    err = hipMemcpy(factor_cpu, factor_gpu_in, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        printf("Failed to copy vector factor_gpu_in from device to host!\n");
        exit(EXIT_FAILURE);
    }

    print_accumulate_point_data (NULL, NULL, NULL, energy_cpu, factor_cpu, NULL, NULL, number_of_points);

    ////////////////////////////
    // Free Memory CPU
    free(energy_cpu);
    free(factor_cpu);

    ///////////////////////////
    // Free memory GPU
    hipFree (point_weights_gpu_in);
    hipFree (partial_density_gpu_in);
    hipFree (partial_density_gpu_accum);
    hipFree (energy_gpu_in);
    hipFree (factor_gpu_in);
    hipFree (dxyz_gpu_in);
    hipFree (dd1_gpu_in);
    hipFree (dd2_gpu_in);
    hipFree (dxyz_gpu_accum);
    hipFree (dd1_gpu_accum);
    hipFree (dd2_gpu_accum);
}


////////////////////////////////////////////////////////////////
// Exchange correlation for DOUBLES

//////////////////////////////////////////////////////////////////
// do_libxc_exchange_correlation_gpu
//
// Simulate a run that compute the exchange correlation functional
// using the GPU version of the LibxcProxy component.
//

void do_libxc_exchange_correlation_gpu (int number_of_points,
    double *dens_cpu,
    double *contracted_gradient_cpu,
    G2G::vec_type<double,4>* grad,
    G2G::vec_type<double,4>* hess1,
    G2G::vec_type<double,4>* hess2) {

    printf("do_libxc_exchange_correlation_gpu(%i)\n", number_of_points);

    /////////////////////////////
    // CUDA ARRAYS
    //template<class scalar_type, bool compute_energy, bool compute_factor, bool lda>
    //void libxc_exchange_correlation_gpu (LibxcProxy<scalar_type, WIDTH>* libxcProxy,

    double* energy_gpu = NULL;
    hipMalloc((void**)&energy_gpu, sizeof(double)*number_of_points);

    double* factor_gpu = NULL;
    hipMalloc ((void**)&factor_gpu, sizeof(double)*number_of_points);

    double* accumulated_density_gpu = NULL;
    hipMalloc ((void**)&accumulated_density_gpu, sizeof(double)*number_of_points);

    double* contracted_gradient_gpu = NULL;
    hipMalloc ((void**)&contracted_gradient_gpu, sizeof(double)*number_of_points);

    G2G::vec_type<double,WIDTH>* dxyz_gpu = NULL;
    hipMalloc((void**)&dxyz_gpu, sizeof(G2G::vec_type<double,4>)*number_of_points);

    G2G::vec_type<double,WIDTH>* dd1_gpu = NULL;
    hipMalloc((void**)&dd1_gpu, sizeof(G2G::vec_type<double,4>)*number_of_points);

    G2G::vec_type<double,WIDTH>* dd2_gpu = NULL;
    hipMalloc((void**)&dd2_gpu, sizeof(G2G::vec_type<double,4>)*number_of_points);

    //////////////////////////////
    // SET CUDA ARRAYS VALUES
    hipMemset(energy_gpu, 0, sizeof(double)*number_of_points);
    hipMemset(factor_gpu, 0, sizeof(double)*number_of_points);
    hipMemset(accumulated_density_gpu, 0, sizeof(double)*number_of_points);
    hipMemset(contracted_gradient_gpu, 0, sizeof(double)*number_of_points);

    hipMemcpy(accumulated_density_gpu, dens_cpu, sizeof(double)*number_of_points, hipMemcpyHostToDevice);
    hipMemcpy(contracted_gradient_gpu, contracted_gradient_cpu, sizeof(double)*number_of_points, hipMemcpyHostToDevice);

    hipMemcpy(dxyz_gpu, grad, sizeof(G2G::vec_type<double,4>)*number_of_points, hipMemcpyHostToDevice);
    hipMemcpy(dd1_gpu, hess1, sizeof(G2G::vec_type<double,4>)*number_of_points, hipMemcpyHostToDevice);
    hipMemcpy(dd2_gpu, hess2, sizeof(G2G::vec_type<double,4>)*number_of_points, hipMemcpyHostToDevice);

    //////////////////////////////
    // CREATE THE PROXY
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<double,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    //////////////////////////////
    // MAKE THE CALLS
    libxc_exchange_correlation_gpu<double, true, true, false> (
	&libxcProxy,
	energy_gpu,
	factor_gpu,
	number_of_points,
	accumulated_density_gpu,
	dxyz_gpu,
        dd1_gpu,
	dd2_gpu);

    /////////////////////////////
    // PRINT THE RESULTS
    double* energy_cpu = (double*)malloc(sizeof(double)*number_of_points);
    double* factor_cpu = (double*)malloc(sizeof(double)*number_of_points);

    memset(energy_cpu,0,sizeof(double)*number_of_points);
    memset(factor_cpu,0,sizeof(double)*number_of_points);

    hipMemcpy(energy_cpu, energy_gpu, sizeof(double)*number_of_points, hipMemcpyDeviceToHost);
    hipMemcpy(factor_cpu, factor_gpu, sizeof(double)*number_of_points, hipMemcpyDeviceToHost);

    print_accumulate_point_data (NULL, NULL, NULL, energy_cpu, factor_cpu, NULL, NULL, number_of_points);

    /////////////////////////////
    // FREE MEMORY
    hipFree(energy_gpu);
    hipFree(factor_gpu);
    hipFree(accumulated_density_gpu);
    hipFree(contracted_gradient_gpu);
    hipFree(dxyz_gpu);
    hipFree(dd1_gpu);
    hipFree(dd2_gpu);

    free(energy_cpu);
    free(factor_cpu);
}

//////////////////////////////////////////////////////////////////
// Test: accumulate_data_for_libxc_test0010
//
// Runs a exchange correlation simulation using the
// GPU version of the LibxcProxy component.
//
void accumulate_data_for_libxc_test0010()
{
    printf("accumulate_data_for_libxc_test0010()\n");

    ////////////////////////////////
    // PARAMS SETUP

    int number_of_points[9] = {221,227,256,537,1796,4007,2910,2910,3492};
    double* dens_cpu[9] = {dens_221,dens_227,dens_256,dens_537,dens_1796,dens_4007,dens_2910_1,dens_2910_2,dens_3492};
    double* contracted_gradients_cpu[9] = {contracted_grad_221,contracted_grad_227,contracted_grad_256,contracted_grad_537,contracted_grad_1796,contracted_grad_4007,contracted_grad_2910_1,contracted_grad_2910_2,contracted_grad_3492};
    G2G::vec_type<double,4>* grads[9] = {grad_221,grad_227,grad_256,grad_537,grad_1796,grad_4007,grad_2910_1,grad_2910_2,grad_3492};
    G2G::vec_type<double,4>* hess1s[9] = {hess1_221,hess1_227,hess1_256,hess1_537,hess1_1796,hess1_4007,hess1_2910_1,hess1_2910_2,hess1_3492};
    G2G::vec_type<double,4>* hess2s[9] = {hess2_221,hess2_227,hess2_256,hess2_537,hess2_1796,hess2_4007,hess2_2910_1,hess2_2910_2,hess2_3492};

    for (int i=0; i<1; i++) {
        do_libxc_exchange_correlation_gpu (number_of_points[i], 
	    dens_cpu[i], 
	    contracted_gradients_cpu[i],
	    grads[i],
	    hess1s[i],
	    hess2s[i]);
    }
}

//////////////////////////////////////////////////////////////////
// do_libxc_exchange_correlation_gpu_floa
//
// Helper function to simulate the Exchange Correlation
// calculations for Floats
//
void do_libxc_exchange_correlation_gpu_float (int number_of_points,
    float *dens_cpu,
    float *contracted_gradient_cpu,
    G2G::vec_type<float,4>* grad,
    G2G::vec_type<float,4>* hess1,
    G2G::vec_type<float,4>* hess2) {

    printf("do_libxc_exchange_correlation_gpu_float(%i)\n", number_of_points);

    /////////////////////////////
    // CUDA ARRAYS
    float* energy_gpu = NULL;
    hipMalloc((void**)&energy_gpu, sizeof(float)*number_of_points);

    float* factor_gpu = NULL;
    hipMalloc ((void**)&factor_gpu, sizeof(float)*number_of_points);

    float* accumulated_density_gpu = NULL;
    hipMalloc ((void**)&accumulated_density_gpu, sizeof(float)*number_of_points);

    float* contracted_gradient_gpu = NULL;
    hipMalloc ((void**)&contracted_gradient_gpu, sizeof(float)*number_of_points);

    G2G::vec_type<float,WIDTH>* dxyz_gpu = NULL;
    hipMalloc((void**)&dxyz_gpu, sizeof(G2G::vec_type<float,4>)*number_of_points);

    G2G::vec_type<float,WIDTH>* dd1_gpu = NULL;
    hipMalloc((void**)&dd1_gpu, sizeof(G2G::vec_type<float,4>)*number_of_points);

    G2G::vec_type<float,WIDTH>* dd2_gpu = NULL;
    hipMalloc((void**)&dd2_gpu, sizeof(G2G::vec_type<float,4>)*number_of_points);

    //////////////////////////////
    // SET CUDA ARRAYS VALUES
    hipMemset(energy_gpu, 0, sizeof(float)*number_of_points);
    hipMemset(factor_gpu, 0, sizeof(float)*number_of_points);
    hipMemset(accumulated_density_gpu, 0, sizeof(float)*number_of_points);
    hipMemset(contracted_gradient_gpu, 0, sizeof(float)*number_of_points);

    hipMemcpy(accumulated_density_gpu, dens_cpu, sizeof(float)*number_of_points, hipMemcpyHostToDevice);
    hipMemcpy(contracted_gradient_gpu, contracted_gradient_cpu, sizeof(float)*number_of_points, hipMemcpyHostToDevice);

    hipMemcpy(dxyz_gpu, grad, sizeof(G2G::vec_type<float,4>)*number_of_points, hipMemcpyHostToDevice);
    hipMemcpy(dd1_gpu, hess1, sizeof(G2G::vec_type<float,4>)*number_of_points, hipMemcpyHostToDevice);
    hipMemcpy(dd2_gpu, hess2, sizeof(G2G::vec_type<float,4>)*number_of_points, hipMemcpyHostToDevice);

    //////////////////////////////
    // CREATE THE PROXY
    const int nspin = 1;
    const int functionalExchange = 1101;
    const int functionalCorrelation = 1130;
    LibxcProxy<float,4> libxcProxy(functionalExchange, functionalCorrelation, nspin);

    //////////////////////////////
    // MAKE THE CALLS
    libxc_exchange_correlation_gpu<float, true, true, false> (
	&libxcProxy,
	energy_gpu,
	factor_gpu,
	number_of_points,
	accumulated_density_gpu,
	dxyz_gpu,
        dd1_gpu,
	dd2_gpu);

    /////////////////////////////
    // PRINT THE RESULTS
    float* energy_cpu = (float*)malloc(sizeof(float)*number_of_points);
    float* factor_cpu = (float*)malloc(sizeof(float)*number_of_points);

    memset(energy_cpu,0,sizeof(float)*number_of_points);
    memset(factor_cpu,0,sizeof(float)*number_of_points);

    hipMemcpy(energy_cpu, energy_gpu, sizeof(float)*number_of_points, hipMemcpyDeviceToHost);
    hipMemcpy(factor_cpu, factor_gpu, sizeof(float)*number_of_points, hipMemcpyDeviceToHost);

    print_accumulate_point_data (NULL, NULL, NULL, energy_cpu, factor_cpu, NULL, NULL, number_of_points);

    /////////////////////////////
    // FREE MEMORY
    hipFree(energy_gpu);
    hipFree(factor_gpu);
    hipFree(accumulated_density_gpu);
    hipFree(contracted_gradient_gpu);
    hipFree(dxyz_gpu);
    hipFree(dd1_gpu);
    hipFree(dd2_gpu);

    free(energy_cpu);
    free(factor_cpu);
}


////////////////////////////////////////////////
// Test: accumulate_data_for_libxc_test0011
//
// Accumulate data test for floats
//
void accumulate_data_for_libxc_test0011()
{
    printf("accumulate_data_for_libxc_test0011()\n");

    ////////////////////////////////
    // PARAMS SETUP

    int number_of_points[9] = {221,227,256,537,1796,4007,2910,2910,3492};
    float* dens_cpu[9] = {dens_221_f,dens_227_f,dens_256_f,dens_537_f,dens_1796_f,dens_4007_f,dens_2910_1_f,dens_2910_2_f,dens_3492_f};
    float* contracted_gradients_cpu[9] = {contracted_grad_221_f,contracted_grad_227_f,contracted_grad_256_f,contracted_grad_537_f,contracted_grad_1796_f,contracted_grad_4007_f,contracted_grad_2910_1_f,contracted_grad_2910_2_f,contracted_grad_3492_f};
    G2G::vec_type<float,4>* grads[9] = {grad_221_f,grad_227_f,grad_256_f,grad_537_f,grad_1796_f,grad_4007_f,grad_2910_1_f,grad_2910_2_f,grad_3492_f};
    G2G::vec_type<float,4>* hess1s[9] = {hess1_221_f,hess1_227_f,hess1_256_f,hess1_537_f,hess1_1796_f,hess1_4007_f,hess1_2910_1_f,hess1_2910_2_f,hess1_3492_f};
    G2G::vec_type<float,4>* hess2s[9] = {hess2_221_f,hess2_227_f,hess2_256_f,hess2_537_f,hess2_1796_f,hess2_4007_f,hess2_2910_1_f,hess2_2910_2_f,hess2_3492_f};

    for (int i=0; i<9; i++) {
        do_libxc_exchange_correlation_gpu_float (number_of_points[i], 
	    dens_cpu[i], 
	    contracted_gradients_cpu[i],
	    grads[i],
	    hess1s[i],
	    hess2s[i]);
    }
}


/////////////////////////////////////
//// MAIN

int main(int argc, char **argv)
{
    printf("*********************\n");
    printf("** Iteration test  **\n");
    printf("*********************\n");

    try {
        iteration_test0010();
    } catch (int e) {
	printf("An exception occurred. Exception Nr. %u \n", e);
	exit (EXIT_FAILURE);
    }

    printf("*************************\n");
    printf("**      Test End       **\n");
    printf("*************************\n");

    return 0;
}